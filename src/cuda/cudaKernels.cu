#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdio.h>
#include "cudaMisc.h"
#include "cudaCppWrappers.h"
#include "injector.hpp"
#include "we_op.hpp"

#define BLOCK_SIZE 16

__constant__ static data_t bnd_coef1[24] = {-24.0/17,59.0/34,-4.0/17,-3.0/34,0,0,-0.5,0,0.5,0,0,0,4.0/43,-59.0/86,0,59.0/86,-4.0/43,0,3.0/98,0,-59.0/98,0,32.0/49,-4.0/49};
__constant__ static data_t coef[10] = {-3.0/4, -5.0/6, -1.0/24, 1.0/6, 1.0/2, 1.0/2, 1.0/6, -1.0/8, 1.0/6, -1.0/8};
__constant__ static data_t bnd_coef2[384] = {920.0/289,-59.0/68,-7549318.0/34157643,-17440994.0/184112825,0,0,0,0,-1740.0/289,0,295314153.0/366719282,262545878.0/1218454145,0,0,0,0,1128.0/289,59.0/68,-19250923.0/26254840,-12192537.0/324892213,0,0,0,0,-308.0/289,0,42283069.0/254173229,-43013531.0/427521546,0,0,0,0,0,0,-18700293.0/1355757959,18700293.0/1355757959,0,0,0,0,0,0,-3.0/833,3.0/833,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
    12.0/17,0,89562243.0/385991318,16914691.0/272440171,0,0,0,0,-59.0/68,0,-47979680.0/48852831,-18034913.0/120051851,0,0,0,0,2.0/17,0,299262497.0/373256703,16156647.0/200473586,0,0,0,0,3.0/68,0,-14723969.0/177744748,22633571.0/584543661,0,0,0,0,0,0,46802031.0/1628311862,-46802031.0/1628311862,0,0,0,0,0,0,441.0/181507,-441.0/181507,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
    -96.0/731,59.0/172,-47632751.0/164317197,-5570723.0/375470930,0,0,0,0,118.0/731,0,54282598.0/49343777,23802793.0/215253532,0,0,0,0,-16.0/731,-59.0/172,-39119273.0/25083370,-35971870.0/61324629,-26254.0/557679,0,0,0,-6.0/731,0,360454121.0/368940022,17254963.0/80047776,1500708.0/7993399,0,0,0,0,0,-18024731.0/79673021,24178273.0/88099647,-26254.0/185893,0,0,0,0,0,-870707.0/620833782,960119.0/1147305747,13564.0/23980197,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
    -36.0/833,0,54741803.0/948483020,-13602043.0/389676498,0,0,0,0,177.0/3332,0,-35820026.0/359121865,24921773.0/534548210,0,0,0,0,-6.0/833,0,813284372.0/948584067,30057666.0/158897885,1500708.0/9108757,0,0,0,-9.0/3332,0,-95056924.0/90903639,-23417695.0/47008088,-7476412.0/9108757,-2.0/49,0,0,0,0,23159719.0/99948527,110687545.0/265515812,4502124.0/9108757,8.0/49,0,0,0,0,-3671038.0/2687426923,-1063649.0/8893843,1473580.0/9108757,-6.0/49,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
    0,0,-9437957.0/1931986386,9437957.0/1931986386,0,0,0,0,0,0,17289851.0/489388053,-17289851.0/489388053,0,0,0,0,0,0,-66355919.0/327412264,24178273.0/98343792,-564461.0/4461432,0,0,0,0,0,17638343.0/74566894,103749401.0/243793650,375177.0/743572,1.0/6,0,0,0,0,-19321801.0/295845927,-50677283.0/62943042,-280535.0/371786,-5.0/6,-1.0/24,0,0,0,5130905.0/5183662662,35039615.0/213452232,1118749.0/2230716,1.0/2,1.0/6,0,0,0,0,0,-1.0/8,1.0/6,-1.0/8,0,0,0,0,0,0,0,0,0,
    0,0,-1.0/784,1.0/784,0,0,0,0,0,0,8673.0/2904112,-8673.0/2904112,0,0,0,0,0,0,-403062.0/320810033,960119.0/1280713392,3391.0/6692148,0,0,0,0,0,-1920494.0/1377228165,-1063649.0/8712336,368395.0/2230716,-1.0/8,0,0,0,0,5130905.0/5183662662,35039615.0/213452232,1118749.0/2230716,1.0/2,1.0/6,0,0,0,-117380.0/2351569839,-3290636.0/80044587,-5580181.0/6692148,-3.0/4,-5.0/6,-1.0/24,0,0,0,0,1.0/6,1.0/2,1.0/2,1.0/6,0,0,0,0,0,-1.0/8,1.0/6,-1.0/8
};

__global__ void cudaDz_interior(bool add, const data_t* in, data_t* out, int nx, int nz, data_t d){
	
    int nc1=4;

    int ix = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    int iz = threadIdx.y + blockIdx.y*BLOCK_SIZE + nc1;
    int blockx = BLOCK_SIZE;
    int blockz = BLOCK_SIZE+4;  // shared block size with halo
    int ixL = threadIdx.x;
    int izL = threadIdx.y;
    int i1=ix*nz;

    data_t coef0=2.0/3;
    data_t coef1=-1.0/12;
    data_t  scale=1.0/d;

    if (blockIdx.x == gridDim.x-1) blockx = nx-BLOCK_SIZE*(gridDim.x-1); // rightmost block
    if (blockIdx.y == gridDim.y-1) blockz = nz-2*nc1-BLOCK_SIZE*(gridDim.y-1)+4; // bottommost block

    __shared__ data_t sh_in[BLOCK_SIZE][BLOCK_SIZE+4]; // allocate shared memory: BLOCKSIZE + halos

    if (ixL<blockx && izL<blockz-4) sh_in[ixL][izL+2] = in[i1+iz]; // copy the stencil tile
    if (ixL<blockx && izL<2) {
        sh_in[ixL][izL] = in[i1+iz-2]; // top halo
        sh_in[ixL][blockz-1-izL] = in[i1+iz+blockz-2-1-2*izL]; // bottom halo
    }
    __syncthreads();

    if (ix < nx && iz < nz - nc1)
        out[ i1+ iz] = add*out[ i1+ iz] + scale * (coef0 * (sh_in[ixL][izL+3] - sh_in[ixL][izL+1]) + coef1 * (sh_in[ixL][izL+4] - sh_in[ixL][izL]));
}

__global__ void cudaDz_bnd(bool add, const data_t* in, data_t* out, int nx, int nz, data_t d){
	
    int nc2=6;

    int ix = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    int iz = threadIdx.y;
    int blockx = BLOCK_SIZE;
    int blockz = 6;  // shared block size with halo
    int ixL = threadIdx.x;
    int i1=ix*nz;

    if (blockIdx.x == gridDim.x-1) blockx = nx-BLOCK_SIZE*(gridDim.x-1); // rightmost block

    __shared__ data_t sh_in[BLOCK_SIZE][6]; // allocate shared memory: BLOCKSIZE + halos

    if (blockIdx.y == 0) // top boundary
    {
        if (ixL<blockx) {
            sh_in[ixL][iz] = in[i1+iz]; // copy the stencil tile
            if (iz<2) sh_in[ixL][blockz-1-iz] = in[i1+blockz-1-iz]; // bottom halo
        }
        __syncthreads();

        if (ix<nx) out[i1+iz] = add* out[i1+iz] + (bnd_coef1[iz*nc2] * sh_in[ixL][0] + bnd_coef1[iz*nc2+1] * sh_in[ixL][1] + bnd_coef1[iz*nc2+2] * sh_in[ixL][2] + bnd_coef1[iz*nc2+3] * sh_in[ixL][3] + bnd_coef1[iz*nc2+4] * sh_in[ixL][4] + bnd_coef1[iz*nc2+5] * sh_in[ixL][5]) / d;
    }
    else  // bottom boundary
    {
        if (ixL<blockx) {
            sh_in[ixL][iz] = in[i1+nz-1-iz]; // copy the stencil tile
            if (iz<2) sh_in[ixL][blockz-1-iz] = in[i1+nz-blockz+iz]; // top halo
        }
        __syncthreads();

        if (ix<nx) out[i1+nz-1-iz] = add* out[i1+nz-1-iz] + (-bnd_coef1[iz*nc2] * sh_in[ixL][0] - bnd_coef1[iz*nc2+1] * sh_in[ixL][1] - bnd_coef1[iz*nc2+2] * sh_in[ixL][2] - bnd_coef1[iz*nc2+3] * sh_in[ixL][3] - bnd_coef1[iz*nc2+4] * sh_in[ixL][4] - bnd_coef1[iz*nc2+5] * sh_in[ixL][5]) / d;
    }
}

__global__ void cudaDx_interior(bool add, const data_t* in, data_t* out, int nx, int nz, data_t d){
	
    int nc1=4;

    int ix = threadIdx.x + blockIdx.x*BLOCK_SIZE + nc1;
    int iz = threadIdx.y + blockIdx.y*BLOCK_SIZE;
    int blockx = BLOCK_SIZE + 4; // shared block size with halo
    int blockz = BLOCK_SIZE;  
    int ixL = threadIdx.x;
    int izL = threadIdx.y;
    int i1=ix*nz;

    data_t coef0=2.0/3;
    data_t coef1=-1.0/12;
    data_t  scale=1.0/d;

    if (blockIdx.x == gridDim.x-1) blockx = nx-2*nc1-BLOCK_SIZE*(gridDim.x-1)+4; // rightmost block
    if (blockIdx.y == gridDim.y-1) blockz = nz-BLOCK_SIZE*(gridDim.y-1); // bottommost block

    __shared__ data_t sh_in[BLOCK_SIZE+4][BLOCK_SIZE]; // allocate shared memory: BLOCKSIZE + halos

    if (ixL<blockx-4 && izL<blockz) sh_in[ixL+2][izL] = in[i1+iz]; // copy the stencil tile

    if (izL<blockz && ixL<2) {
        sh_in[ixL][izL] = in[(ix-2)*nz+iz]; // left halo
        sh_in[blockx-1-ixL][izL] = in[(ix+blockx-2-1-2*ixL)*nz+iz]; // right halo
    }
    __syncthreads();

    if (ix < nx-nc1 && iz < nz)
        out[ i1+ iz] = add*out[ i1+ iz] + scale * (coef0 * (sh_in[ixL+3][izL] - sh_in[ixL+1][izL]) + coef1 * (sh_in[ixL+4][izL] - sh_in[ixL][izL]));
}

__global__ void cudaDx_bnd(bool add, const data_t* in, data_t* out, int nx, int nz, data_t d){
	
    int nc2=6;

    int ix = threadIdx.x;
    int iz = threadIdx.y + blockIdx.y*BLOCK_SIZE;
    int blockx = 6; // shared block size with halo
    int blockz = BLOCK_SIZE;  
    int izL = threadIdx.y;
    int i1=ix*nz;

    if (blockIdx.y == gridDim.y-1) blockz = nz-BLOCK_SIZE*(gridDim.y-1); // bottommost block

    __shared__ data_t sh_in[6][BLOCK_SIZE]; // allocate shared memory: BLOCKSIZE + halos

    if (blockIdx.x == 0) // left boundary
    {
        if (izL<blockz) {
            sh_in[ix][izL] = in[i1+iz]; // copy the stencil tile
            if (ix<2) sh_in[blockx-1-ix][izL] = in[(blockx-1-ix)*nz+iz]; // right halo
        }
        __syncthreads();

        if (iz<nz) out[i1+iz] = add* out[i1+iz] + (bnd_coef1[ix*nc2] * sh_in[0][izL] + bnd_coef1[ix*nc2+1] * sh_in[1][izL] + bnd_coef1[ix*nc2+2] * sh_in[2][izL] + bnd_coef1[ix*nc2+3] * sh_in[3][izL] + bnd_coef1[ix*nc2+4] * sh_in[4][izL] + bnd_coef1[ix*nc2+5] * sh_in[5][izL]) / d; 
    }
    else  // right boundary
    {
        if (izL<blockz) {
            sh_in[ix][izL] = in[(nx-1-ix)*nz+iz]; // copy the stencil tile
            if (ix<2) sh_in[blockx-1-ix][izL] = in[(nx-blockx+ix)*nz+iz]; // left halo
        }
        __syncthreads();

        i1=(nx-1-ix)*nz;
        if (iz<nz) out[i1+iz] = add* out[i1+iz] + (-bnd_coef1[ix*nc2] * sh_in[0][izL] - bnd_coef1[ix*nc2+1] * sh_in[1][izL] - bnd_coef1[ix*nc2+2] * sh_in[2][izL] - bnd_coef1[ix*nc2+3] * sh_in[3][izL] - bnd_coef1[ix*nc2+4] * sh_in[4][izL] - bnd_coef1[ix*nc2+5] * sh_in[5][izL]) / d;
    }
}

__global__ void cudaMultDz_interior(bool add, const data_t* in, data_t* out, int nx, int nz, data_t d, const data_t* par, data_t a){
	
    int nc1=4;

    int ix = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    int iz = threadIdx.y + blockIdx.y*BLOCK_SIZE + nc1;
    int blockx = BLOCK_SIZE;
    int blockz = BLOCK_SIZE+4;  // shared block size with halo
    int ixL = threadIdx.x;
    int izL = threadIdx.y;
    int i1=ix*nz;

    data_t coef0=2.0/3;
    data_t coef1=-1.0/12;
    data_t  scale=a/d;

    if (blockIdx.x == gridDim.x-1) blockx = nx-BLOCK_SIZE*(gridDim.x-1); // rightmost block
    if (blockIdx.y == gridDim.y-1) blockz = nz-2*nc1-BLOCK_SIZE*(gridDim.y-1)+4; // bottommost block

    __shared__ data_t sh_in[BLOCK_SIZE][BLOCK_SIZE+4]; // allocate shared memory: BLOCKSIZE + halos
    __shared__ data_t sh_par[BLOCK_SIZE][BLOCK_SIZE+4];
    if (ixL<blockx && izL<blockz-4) {
        sh_in[ixL][izL+2] = in[i1+iz]; // copy the stencil tile
        sh_par[ixL][izL+2] = par[i1+iz];
    }
    if (ixL<blockx && izL<2) {
        sh_in[ixL][izL] = in[i1+iz-2]; // top halo
        sh_par[ixL][izL] = par[i1+iz-2]; 
        sh_in[ixL][blockz-1-izL] = in[i1+iz+blockz-2-1-2*izL]; // bottom halo
        sh_par[ixL][blockz-1-izL] = par[i1+iz+blockz-2-1-2*izL];
    }
    __syncthreads();

    if (ix < nx && iz < nz - nc1)
        out[ i1+ iz] = add*out[ i1+ iz] + scale * (coef0 * (sh_in[ixL][izL+3]*sh_par[ixL][izL+3] - sh_in[ixL][izL+1]*sh_par[ixL][izL+1]) + coef1 * (sh_in[ixL][izL+4]*sh_par[ixL][izL+4] - sh_in[ixL][izL]*sh_par[ixL][izL]));
}

__global__ void cudaMultDz_bnd(bool add, const data_t* in, data_t* out, int nx, int nz, data_t d, const data_t* par, data_t a){
	
    int nc2=6;

    int ix = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    int iz = threadIdx.y;
    int blockx = BLOCK_SIZE;
    int blockz = 6;  // shared block size with halo
    int ixL = threadIdx.x;
    int i1=ix*nz;

    if (blockIdx.x == gridDim.x-1) blockx = nx-BLOCK_SIZE*(gridDim.x-1); // rightmost block

    __shared__ data_t sh_in[BLOCK_SIZE][6]; // allocate shared memory: BLOCKSIZE + halos
    __shared__ data_t sh_par[BLOCK_SIZE][6];

    if (blockIdx.y == 0) // top boundary
    {
        if (ixL<blockx) {
            sh_in[ixL][iz] = in[i1+iz]; // copy the stencil tile
            sh_par[ixL][iz] = par[i1+iz];
            if (iz<2) {
                sh_in[ixL][blockz-1-iz] = in[i1+blockz-1-iz]; // bottom halo
                sh_par[ixL][blockz-1-iz] = par[i1+blockz-1-iz]; 
            }
        }
        __syncthreads();

        if (ix<nx) out[i1+iz] = add* out[i1+iz] + a/d * (bnd_coef1[iz*nc2] * sh_in[ixL][0]*sh_par[ixL][0] + bnd_coef1[iz*nc2+1] * sh_in[ixL][1]*sh_par[ixL][1] + bnd_coef1[iz*nc2+2] * sh_in[ixL][2]*sh_par[ixL][2] + bnd_coef1[iz*nc2+3] * sh_in[ixL][3]*sh_par[ixL][3] + bnd_coef1[iz*nc2+4] * sh_in[ixL][4]*sh_par[ixL][4] + bnd_coef1[iz*nc2+5] * sh_in[ixL][5]*sh_par[ixL][5]);
    }
    else  // bottom boundary
    {
        if (ixL<blockx) {
            sh_in[ixL][iz] = in[i1+nz-1-iz]; // copy the stencil tile
            sh_par[ixL][iz] = par[i1+nz-1-iz];
            if (iz<2) {
                sh_in[ixL][blockz-1-iz] = in[i1+nz-blockz+iz]; // top halo
                sh_par[ixL][blockz-1-iz] = par[i1+nz-blockz+iz];
            }
        }
        __syncthreads();

        if (ix<nx) out[i1+nz-1-iz] = add* out[i1+nz-1-iz] + a/d * (-bnd_coef1[iz*nc2] * sh_in[ixL][0]*sh_par[ixL][0] - bnd_coef1[iz*nc2+1] * sh_in[ixL][1]*sh_par[ixL][1] - bnd_coef1[iz*nc2+2] * sh_in[ixL][2]*sh_par[ixL][2] - bnd_coef1[iz*nc2+3] * sh_in[ixL][3]*sh_par[ixL][3] - bnd_coef1[iz*nc2+4] * sh_in[ixL][4]*sh_par[ixL][4] - bnd_coef1[iz*nc2+5] * sh_in[ixL][5]*sh_par[ixL][5]);
       
    }
}

__global__ void cudaMultDx_interior(bool add, const data_t* in, data_t* out, int nx, int nz, data_t d, const data_t* par, data_t a){
	
    int nc1=4;

    int ix = threadIdx.x + blockIdx.x*BLOCK_SIZE + nc1;
    int iz = threadIdx.y + blockIdx.y*BLOCK_SIZE;
    int blockx = BLOCK_SIZE + 4; // shared block size with halo
    int blockz = BLOCK_SIZE;  
    int ixL = threadIdx.x;
    int izL = threadIdx.y;
    int i1=ix*nz;

    data_t coef0=2.0/3;
    data_t coef1=-1.0/12;
    data_t  scale=a/d;

    if (blockIdx.x == gridDim.x-1) blockx = nx-2*nc1-BLOCK_SIZE*(gridDim.x-1)+4; // rightmost block
    if (blockIdx.y == gridDim.y-1) blockz = nz-BLOCK_SIZE*(gridDim.y-1); // bottommost block

    __shared__ data_t sh_in[BLOCK_SIZE+4][BLOCK_SIZE]; // allocate shared memory: BLOCKSIZE + halos
    __shared__ data_t sh_par[BLOCK_SIZE+4][BLOCK_SIZE];
    if (ixL<blockx-4 && izL<blockz) {
        sh_in[ixL+2][izL] = in[i1+iz]; // copy the stencil tile
        sh_par[ixL+2][izL] = par[i1+iz];
    }

    if (izL<blockz && ixL<2) {
        sh_in[ixL][izL] = in[(ix-2)*nz+iz]; // left halo
        sh_par[ixL][izL] = par[(ix-2)*nz+iz];
        sh_in[blockx-1-ixL][izL] = in[(ix+blockx-2-1-2*ixL)*nz+iz]; // right halo
        sh_par[blockx-1-ixL][izL] = par[(ix+blockx-2-1-2*ixL)*nz+iz];
    }
    __syncthreads();

    if (ix < nx-nc1 && iz < nz)
        out[ i1+ iz] = add*out[ i1+ iz] + scale * (coef0 * (sh_in[ixL+3][izL]*sh_par[ixL+3][izL] - sh_in[ixL+1][izL]*sh_par[ixL+1][izL]) + coef1 * (sh_in[ixL+4][izL]*sh_par[ixL+4][izL] - sh_in[ixL][izL]*sh_par[ixL][izL]));
}

__global__ void cudaMultDx_bnd(bool add, const data_t* in, data_t* out, int nx, int nz, data_t d, const data_t* par, data_t a){
	
    int nc2=6;

    int ix = threadIdx.x;
    int iz = threadIdx.y + blockIdx.y*BLOCK_SIZE;
    int blockx = 6; // shared block size with halo
    int blockz = BLOCK_SIZE;  
    int izL = threadIdx.y;
    int i1=ix*nz;

    if (blockIdx.y == gridDim.y-1) blockz = nz-BLOCK_SIZE*(gridDim.y-1); // bottommost block

    __shared__ data_t sh_in[6][BLOCK_SIZE]; // allocate shared memory: BLOCKSIZE + halos
    __shared__ data_t sh_par[6][BLOCK_SIZE];

    if (blockIdx.x == 0) // left boundary
    {
        if (izL<blockz) {
            sh_in[ix][izL] = in[i1+iz]; // copy the stencil tile
            sh_par[ix][izL] = par[i1+iz]; 
            if (ix<2){
                sh_in[blockx-1-ix][izL] = in[(blockx-1-ix)*nz+iz]; // right halo
                sh_par[blockx-1-ix][izL] = par[(blockx-1-ix)*nz+iz];
            }
        }
        __syncthreads();

        if (iz<nz) out[i1+iz] = add* out[i1+iz] + a/d * (bnd_coef1[ix*nc2] * sh_in[0][izL]*sh_par[0][izL] + bnd_coef1[ix*nc2+1] * sh_in[1][izL]*sh_par[1][izL] + bnd_coef1[ix*nc2+2] * sh_in[2][izL]*sh_par[2][izL] + bnd_coef1[ix*nc2+3] * sh_in[3][izL]*sh_par[3][izL] + bnd_coef1[ix*nc2+4] * sh_in[4][izL]*sh_par[4][izL] + bnd_coef1[ix*nc2+5] * sh_in[5][izL]*sh_par[5][izL]); 
    }
    else  // right boundary
    {
        if (izL<blockz) {
            sh_in[ix][izL] = in[(nx-1-ix)*nz+iz]; // copy the stencil tile
            sh_par[ix][izL] = par[(nx-1-ix)*nz+iz];
            if (ix<2) {
                sh_in[blockx-1-ix][izL] = in[(nx-blockx+ix)*nz+iz]; // left halo
                sh_par[blockx-1-ix][izL] = par[(nx-blockx+ix)*nz+iz];
            }
        }
        __syncthreads();

        i1=(nx-1-ix)*nz;
        if (iz<nz) out[i1+iz] = add* out[i1+iz] + a/d * (-bnd_coef1[ix*nc2] * sh_in[0][izL]*sh_par[0][izL] - bnd_coef1[ix*nc2+1] * sh_in[1][izL]*sh_par[1][izL] - bnd_coef1[ix*nc2+2] * sh_in[2][izL]*sh_par[2][izL] - bnd_coef1[ix*nc2+3] * sh_in[3][izL]*sh_par[3][izL] - bnd_coef1[ix*nc2+4] * sh_in[4][izL]*sh_par[4][izL] - bnd_coef1[ix*nc2+5] * sh_in[5][izL]*sh_par[5][izL]);
    }
}

__global__ void cudaDzz_var_interior(bool add, const data_t* in, data_t* out, int nx, int nz, data_t d, const data_t * par, data_t a){
	
    int nc1=6;
    data_t d2 = d*d;

    int ix = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    int iz = threadIdx.y + blockIdx.y*BLOCK_SIZE + nc1;
    int blockx = BLOCK_SIZE;
    int blockz = BLOCK_SIZE+4;  // shared block size with halo
    int ixL = threadIdx.x;
    int izL = threadIdx.y;
    int i1=ix*nz;

    if (blockIdx.x == gridDim.x-1) blockx = nx-BLOCK_SIZE*(gridDim.x-1); // rightmost block
    if (blockIdx.y == gridDim.y-1) blockz = nz-2*nc1-BLOCK_SIZE*(gridDim.y-1)+4; // bottommost block

    __shared__ data_t sh_in[BLOCK_SIZE][BLOCK_SIZE+4]; // allocate shared memory: BLOCKSIZE + halos
    __shared__ data_t sh_par[BLOCK_SIZE][BLOCK_SIZE+4];
    if (ixL<blockx && izL<blockz-4) {
        sh_in[ixL][izL+2] = in[i1+iz]; // copy the stencil tile
        sh_par[ixL][izL+2] = par[i1+iz];
    }

    if (ixL<blockx && izL<2) {
        sh_in[ixL][izL] = in[i1+iz-2]; // top halo
        sh_par[ixL][izL] = par[i1+iz-2];
        sh_in[ixL][blockz-1-izL] = in[i1+iz+blockz-2-1-2*izL]; // bottom halo
        sh_par[ixL][blockz-1-izL] = par[i1+iz+blockz-2-1-2*izL];
    }
    __syncthreads();

    if (ix < nx && iz < nz - nc1)
        out[i1+iz] = add*out[i1+iz]
                                    + a/d2 * ( (coef[0]*sh_par[ixL][izL+2]+coef[1]*(sh_par[ixL][izL+1]+sh_par[ixL][izL+3])+coef[2]*(sh_par[ixL][izL]+sh_par[ixL][izL+4]))*sh_in[ixL][izL+2] 
                                    + (coef[3]*sh_par[ixL][izL]+coef[4]*sh_par[ixL][izL+1]+coef[5]*sh_par[ixL][izL+2]+coef[6]*sh_par[ixL][izL+3])*sh_in[ixL][izL+1]
                                    + (coef[3]*sh_par[ixL][izL+4]+coef[4]*sh_par[ixL][izL+3]+coef[5]*sh_par[ixL][izL+2]+coef[6]*sh_par[ixL][izL+1])*sh_in[ixL][izL+3]
                                    + (coef[7]*sh_par[ixL][izL]+coef[8]*sh_par[ixL][izL+1]+coef[9]*sh_par[ixL][izL+2])*sh_in[ixL][izL]
                                    + (coef[7]*sh_par[ixL][izL+4]+coef[8]*sh_par[ixL][izL+3]+coef[9]*sh_par[ixL][izL+2])*sh_in[ixL][izL+4]
                                    );

}

__global__ void cudaDzz_var_bnd(bool add, const data_t* in, data_t* out, int nx, int nz, data_t d, const data_t * par, data_t a){
	
    int nc2=8, nc3=8;
    data_t  val=0;
    data_t d2 = d*d;

    int ix = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    int iz = threadIdx.y;
    int blockx = BLOCK_SIZE;
    int blockz = 8;  // shared block size with halo
    int ixL = threadIdx.x;
    int i1=ix*nz;

    if (blockIdx.x == gridDim.x-1) blockx = nx-BLOCK_SIZE*(gridDim.x-1); // rightmost block

    __shared__ data_t sh_in[BLOCK_SIZE][8]; // allocate shared memory: BLOCKSIZE + halos
    __shared__ data_t sh_par[BLOCK_SIZE][8];

    if (blockIdx.y == 0) // top boundary
    {
        if (ixL<blockx) {
            sh_in[ixL][iz] = in[i1+iz]; // copy the stencil tile
            sh_par[ixL][iz] = par[i1+iz];
            if (iz<2) {
                sh_in[ixL][blockz-1-iz] = in[i1+blockz-1-iz]; // bottom halo
                sh_par[ixL][blockz-1-iz] = par[i1+blockz-1-iz];
            }
        }
        __syncthreads();

        if (ix<nx)
        {
            out[i1+iz] = add*out[i1+iz];
            int i2=iz*nc2*nc3;
            for (int j = 0; j<nc2; j++){
                val=0;
                for (int k = 0; k<nc3; k++){
                        val += bnd_coef2[i2+j*nc3+k] * sh_par[ixL][k];
                }
                out[i1+iz] += a * val * sh_in[ixL][j] / d2;
            }
        }
    }
    else  // bottom boundary
    {
        if (ixL<blockx) {
            sh_in[ixL][iz] = in[i1+nz-1-iz]; // copy the stencil tile
            sh_par[ixL][iz] = par[i1+nz-1-iz];
            if (iz<2) {
                sh_in[ixL][blockz-1-iz] = in[i1+nz-blockz+iz]; // top halo
                sh_par[ixL][blockz-1-iz] = par[i1+nz-blockz+iz];
            }
        }
        __syncthreads();
        if (ix<nx){
            out[i1+nz-1-iz] = add*out[i1+nz-1-iz];
            int i2=iz*nc2*nc3;
            for (int j = 0; j<nc2; j++){
                    val=0;
                for (int k = 0; k<nc3; k++){
                        val += bnd_coef2[i2+j*nc3+k] * sh_par[ixL][k];
                }
                out[i1+nz-1-iz] +=  a * val * sh_in[ixL][j]/d2;
            }
        }
    }
}

__global__ void cudaDxx_var_interior(bool add, const data_t* in, data_t* out, int nx, int nz, data_t d, const data_t * par, data_t a){
	
    int nc1=6;
    data_t d2 = d*d;

    int ix = threadIdx.x + blockIdx.x*BLOCK_SIZE + nc1;
    int iz = threadIdx.y + blockIdx.y*BLOCK_SIZE;
    int blockx = BLOCK_SIZE + 4; // shared block size with halo
    int blockz = BLOCK_SIZE;  
    int ixL = threadIdx.x;
    int izL = threadIdx.y;
    int i1=ix*nz;

    if (blockIdx.x == gridDim.x-1) blockx = nx-2*nc1-BLOCK_SIZE*(gridDim.x-1)+4; // rightmost block
    if (blockIdx.y == gridDim.y-1) blockz = nz-BLOCK_SIZE*(gridDim.y-1); // bottommost block

    __shared__ data_t sh_in[BLOCK_SIZE+4][BLOCK_SIZE]; // allocate shared memory: BLOCKSIZE + halos
    __shared__ data_t sh_par[BLOCK_SIZE+4][BLOCK_SIZE];
    if (ixL<blockx-4 && izL<blockz) {
        sh_in[ixL+2][izL] = in[i1+iz]; // copy the stencil tile
        sh_par[ixL+2][izL] = par[i1+iz];
    }

    if (izL<blockz && ixL<2) {
        sh_in[ixL][izL] = in[(ix-2)*nz+iz]; // left halo
        sh_par[ixL][izL] = par[(ix-2)*nz+iz];
        sh_in[blockx-1-ixL][izL] = in[(ix+blockx-2-1-2*ixL)*nz+iz]; // right halo
        sh_par[blockx-1-ixL][izL] = par[(ix+blockx-2-1-2*ixL)*nz+iz];
    }
    __syncthreads();

    if (ix < nx-nc1 && iz < nz)
        out[ ix* nz+ iz] = add*out[ ix* nz+ iz] + a/d2 * ( (coef[0]* sh_par[ixL+2][izL]+coef[1]*( sh_par[ixL+1][izL]+ sh_par[ixL+3][izL])+coef[2]*( sh_par[ixL][izL]+ sh_par[ixL+4][izL]))*sh_in[ixL+2][izL] 
                        + (coef[3]* sh_par[ixL][izL]+coef[4]* sh_par[ixL+1][izL]+coef[5]* sh_par[ixL+2][izL]+coef[6]* sh_par[ixL+3][izL])*sh_in[ixL+1][izL]
                        + (coef[3]* sh_par[ixL+4][izL]+coef[4]* sh_par[ixL+3][izL]+coef[5]* sh_par[ixL+2][izL]+coef[6]* sh_par[ixL+1][izL])*sh_in[ixL+3][izL]
                        + (coef[7]* sh_par[ixL][izL]+coef[8]* sh_par[ixL+1][izL]+coef[9]* sh_par[ixL+2][izL])*sh_in[ixL][izL]
                        + (coef[7]* sh_par[ixL+4][izL]+coef[8]* sh_par[ixL+3][izL]+coef[9]* sh_par[ixL+2][izL])*sh_in[ixL+4][izL]
                        );
}

__global__ void cudaDxx_var_bnd(bool add, const data_t* in, data_t* out, int nx, int nz, data_t d, const data_t * par, data_t a){
	
    int nc2=8, nc3=8;
    data_t  val=0;
    data_t d2 = d*d;

    int ix = threadIdx.x;
    int iz = threadIdx.y + blockIdx.y*BLOCK_SIZE;
    int blockx = 8; // shared block size with halo
    int blockz = BLOCK_SIZE;  
    int izL = threadIdx.y;
    int i1=ix*nz;

    if (blockIdx.y == gridDim.y-1) blockz = nz-BLOCK_SIZE*(gridDim.y-1); // bottommost block

    __shared__ data_t sh_in[8][BLOCK_SIZE]; // allocate shared memory: BLOCKSIZE + halos
    __shared__ data_t sh_par[8][BLOCK_SIZE];

    if (blockIdx.x == 0) // left boundary
    {
        if (izL<blockz) {
            sh_in[ix][izL] = in[i1+iz]; // copy the stencil tile
            sh_par[ix][izL] = par[i1+iz];
            if (ix<2) {
                sh_in[blockx-1-ix][izL] = in[(blockx-1-ix)*nz+iz]; // right halo
                sh_par[blockx-1-ix][izL] = par[(blockx-1-ix)*nz+iz];
            }
        }
        __syncthreads();

        if (iz<nz)
        {
            out[i1+iz] = add*out[i1+iz];
            int i2=ix*nc2*nc3;
            for (int j = 0; j<nc2; j++){
                val=0;
                for (int k = 0; k<nc3; k++){
                        val += bnd_coef2[i2+j*nc3+k] * sh_par[k][izL];
                }
                out[i1+iz] += a * val * sh_in[j][izL] / d2;
            }
        }
    }
    else  // right boundary
    {
        if (izL<blockz) {
            sh_in[ix][izL] = in[(nx-1-ix)*nz+iz]; // copy the stencil tile
            sh_par[ix][izL] = par[(nx-1-ix)*nz+iz];
            if (ix<2) {
                sh_in[blockx-1-ix][izL] = in[(nx-blockx+ix)*nz+iz]; // left halo
                sh_par[blockx-1-ix][izL] = par[(nx-blockx+ix)*nz+iz];
            }
        }
        __syncthreads();

        if (iz<nz)
        {
            out[(nx-1-ix)*nz+iz] = add*out[(nx-1-ix)*nz+iz];
            int i2=ix*nc2*nc3;
            for (int j = 0; j<nc2; j++){
                val=0;
                for (int k = 0; k<nc3; k++){
                        val += bnd_coef2[i2+j*nc3+k] * sh_par[k][izL];
                }
                out[(nx-1-ix)*nz+iz] +=  a * val * sh_in[j][izL]/d2;
            }
        }
    }
}

__global__ void cudaEsatNeumannTop(bool add, const data_t* in0, const data_t*in1, data_t* out, int nx, int nz, data_t dx, data_t dz, const data_t * par0, const data_t * par1, data_t a0, data_t a1)
{
    int ix = threadIdx.x + blockIdx.x*BLOCK_SIZE;

    data_t coef[2] = {2.0/3,-1.0/12};
    data_t scoef[4] = {11.0/6, -3, 1.5, -1.0/3};
    data_t h0 = 17.0/48;
    int nc1=4, nc2=6;
    data_t sumx=0, sumz=0;

    // top left
    if (ix<nc1)
    {
        // (Dx.in1)_0
        sumx=0;
        for (int j=0; j<nc2; j++){
            sumx += bnd_coef1[ix*nc2+j] * in0[j*nz];
        }

        // (Sz.in2)_0
        sumz = 0;
        for (int iz = 0; iz < 4; iz++){
            sumz += scoef[iz] * in1[ix*nz+iz];
        }
        out[ix*nz] = add*out[ix*nz] - 1.0 /(dz * h0) * (-a0*par0[ix*nz]*sumx/dx + a1*par1[ix*nz]*sumz/dz);
    }

    // top middle
    else if (ix<nx-nc1)
    {
        // (Dx.in1)_0
        sumx=0;
        for (int j=1; j<=2; j++){
            sumx += coef[j-1] * (in0[(ix+j)*nz]-in0[(ix-j)*nz]);
        }

        // (Sz.in2)_0
        sumz = 0;
        for (int iz = 0; iz < 4; iz++){
            sumz += scoef[iz] * in1[ix*nz+iz];
        }

        out[ix*nz] = add*out[ix*nz] - 1.0 /(dz * h0) * (-a0*par0[ix*nz]*sumx/dx + a1*par1[ix*nz]*sumz/dz);
    }

    // top right
    else if (ix<nx)
    {
        int jx=nx-ix-1;
        // (Dx.in1)_0
        sumx=0;
        for (int j=0; j<nc2; j++){
            sumx -= bnd_coef1[jx*nc2+j] * in0[(nx-1-j)*nz];
        }

        // (Sz.in2)_0
        sumz = 0;
        for (int iz = 0; iz < 4; iz++){
            sumz += scoef[iz] * in1[(nx-1-jx)*nz+iz];
        }
        out[(nx-1-jx)*nz] = add*out[(nx-1-jx)*nz] - 1.0 /(dz * h0) * (-a0*par0[(nx-1-jx)*nz]*sumx/dx + a1*par1[(nx-1-jx)*nz]*sumz/dz);
    }
}

__global__ void cudaEsatNeumannBottom(bool add, const data_t* in0, const data_t*in1, data_t* out, int nx, int nz, data_t dx, data_t dz, const data_t * par0, const data_t * par1, data_t a0, data_t a1)
{
    int ix = threadIdx.x + blockIdx.x*BLOCK_SIZE;

    data_t coef[2] = {2.0/3,-1.0/12};
    data_t scoef[4] = {11.0/6, -3, 1.5, -1.0/3};
    data_t h0 = 17.0/48;
    int nc1=4, nc2=6;
    data_t sumx=0, sumz=0;

    // bottom left
    if (ix<nc1)
    {
        // (Dx.in1)_0
        sumx=0;
        for (int j=0; j<nc2; j++){
            sumx += bnd_coef1[ix*nc2+j] * in0[j*nz+nz-1];
        }

        // (Sz.in2)_0
        sumz = 0;
        for (int iz = 0; iz < 4; iz++){
            sumz += scoef[iz] * in1[ix*nz+nz-1-iz];
        }
        out[ix*nz+nz-1] = add*out[ix*nz+nz-1] - 1.0 /(dz * h0) * (a0*par0[ix*nz+nz-1]*sumx/dx + a1*par1[ix*nz+nz-1]*sumz/dz);
    }

    // bottom middle
    else if (ix<nx-nc1)
    {
        // (Dx.in1)_0
        sumx=0;
        for (int j=1; j<=2; j++){
            sumx += coef[j-1] * (in0[(ix+j)*nz+nz-1]-in0[(ix-j)*nz+nz-1]);
        }

        // (Sz.in2)_0
        sumz = 0;
        for (int iz = 0; iz < 4; iz++){
            sumz += scoef[iz] * in1[ix*nz+nz-1-iz];
        }

        out[ix*nz+nz-1] = add*out[ix*nz+nz-1] - 1.0 /(dz * h0) * (a0*par0[ix*nz+nz-1]*sumx/dx + a1*par1[ix*nz+nz-1]*sumz/dz);
    }

    // bottom right
    else if (ix<nx)
    {
        int jx=nx-ix-1;
        // (Dx.in1)_0
        sumx=0;
        for (int j=0; j<nc2; j++){
            sumx -= bnd_coef1[jx*nc2+j] * in0[(nx-1-j)*nz+nz-1];
        }

        // (Sz.in2)_0
        sumz = 0;
        for (int iz = 0; iz < 4; iz++){
            sumz += scoef[iz] * in1[(nx-1-jx)*nz+nz-1-iz];
        }
        out[(nx-1-jx)*nz+nz-1] = add*out[(nx-1-jx)*nz+nz-1] - 1.0 /(dz * h0) * (a0*par0[(nx-1-jx)*nz+nz-1]*sumx/dx + a1*par1[(nx-1-jx)*nz+nz-1]*sumz/dz);
    }
}

__global__ void cudaEsatNeumannLeft(bool add, const data_t* in0, const data_t*in1, data_t* out, int nx, int nz, data_t dx, data_t dz, const data_t * par0, const data_t * par1, data_t a0, data_t a1)
{
    int iz = threadIdx.y + blockIdx.y*BLOCK_SIZE;

    data_t coef[2] = {2.0/3,-1.0/12};
    data_t scoef[4] = {11.0/6, -3, 1.5, -1.0/3};
    data_t h0 = 17.0/48;
    int nc1=4, nc2=6;
    data_t sumx=0, sumz=0;

    // left top
    if (iz<nc1)
    {
        // (Dz.in1)_0
        sumz=0;
        for (int j=0; j<nc2; j++){
            sumz += bnd_coef1[iz*nc2+j] * in0[j];
        }

        // (Sx.in2)_0
        sumx = 0;
        for (int ix = 0; ix < 4; ix++){
            sumx += scoef[ix] * in1[ix*nz+iz];
        }
        out[iz] = add*out[iz] - 1.0 /(dx * h0) * (-a0*par0[iz]*sumz/dz + a1*par1[iz]*sumx/dx);
    }

    // left middle
    else if (iz<nz-nc1)
    {
        // (Dz.in1)_0
        sumz=0;
        for (int j=1; j<=2; j++){
            sumz += coef[j-1] * (in0[iz+j]-in0[iz-j]);
        }

        // (Sx.in2)_0
        sumx = 0;
        for (int ix = 0; ix < 4; ix++){
            sumx += scoef[ix] * in1[ix*nz+iz];
        }

        out[iz] = add*out[iz] - 1.0 /(dx * h0) * (-a0*par0[iz]*sumz/dz + a1*par1[iz]*sumx/dx);
    }

    // left bottom
    else if (iz<nz)
    {
        int jz=nz-iz-1;
        // (Dz.in1)_0
        sumz=0;
        for (int j=0; j<nc2; j++){
            sumz -= bnd_coef1[jz*nc2+j] * in0[nz-1-j];
        }

        // (Sx.in2)_0
        sumx = 0;
        for (int ix = 0; ix < 4; ix++){
            sumx += scoef[ix] * in1[ix*nz+nz-1-jz];
        }
        out[nz-1-jz] = add*out[nz-1-jz] - 1.0 /(dx * h0) * (-a0*par0[nz-1-jz]*sumz/dz + a1*par1[nz-1-jz]*sumx/dx);
    }
}

__global__ void cudaEsatNeumannRight(bool add, const data_t* in0, const data_t*in1, data_t* out, int nx, int nz, data_t dx, data_t dz, const data_t * par0, const data_t * par1, data_t a0, data_t a1)
{
    int iz = threadIdx.y + blockIdx.y*BLOCK_SIZE;

    data_t coef[2] = {2.0/3,-1.0/12};
    data_t scoef[4] = {11.0/6, -3, 1.5, -1.0/3};
    data_t h0 = 17.0/48;
    int nc1=4, nc2=6;
    data_t sumx=0, sumz=0;

    // right top
    if (iz<nc1)
    {
        // (Dz.in1)_0
        sumz=0;
        for (int j=0; j<nc2; j++){
            sumz += bnd_coef1[iz*nc2+j] * in0[(nx-1)*nz+j];
        }

        // (Sx.in2)_0
        sumx = 0;
        for (int ix = 0; ix < 4; ix++){
            sumx += scoef[ix] * in1[(nx-1-ix)*nz+iz];
        }
        out[(nx-1)*nz+iz] = add*out[(nx-1)*nz+iz] - 1.0 /(dx * h0) * (a0*par0[(nx-1)*nz+iz]*sumz/dz + a1*par1[(nx-1)*nz+iz]*sumx/dx);
    }

    // right middle
    else if (iz<nz-nc1)
    {
        // (Dz.in1)_0
        sumz=0;
        for (int j=1; j<=2; j++){
            sumz += coef[j-1] * (in0[(nx-1)*nz+iz+j]-in0[(nx-1)*nz+iz-j]);
        }

        // (Sx.in2)_0
        sumx = 0;
        for (int ix = 0; ix < 4; ix++){
            sumx += scoef[ix] * in1[(nx-1-ix)*nz+iz];
        }

        out[(nx-1)*nz+iz] = add*out[(nx-1)*nz+iz] - 1.0 /(dx * h0) * (a0*par0[(nx-1)*nz+iz]*sumz/dz + a1*par1[(nx-1)*nz+iz]*sumx/dx);
    }

    // right bottom
    else if (iz<nz)
    {
        int jz=nz-1-iz;
        // (Dz.in1)_0
        sumz=0;
        for (int j=0; j<nc2; j++){
            sumz -= bnd_coef1[jz*nc2+j] * in0[(nx-1)*nz+nz-1-j];
        }

        // (Sx.in2)_0
        sumx = 0;
        for (int ix = 0; ix < 4; ix++){
            sumx += scoef[ix] * in1[(nx-1-ix)*nz+nz-1-jz];
        }
        out[(nx-1)*nz+nz-1-jz] = add*out[(nx-1)*nz+nz-1-jz] - 1.0 /(dx * h0) * (a0*par0[(nx-1)*nz+nz-1-jz]*sumz/dz + a1*par1[(nx-1)*nz+nz-1-jz]*sumx/dx);
    }
}

__global__ void cudaEsatAbsorbingTop(bool add, const data_t* in0, const data_t* in1, const data_t* in2, data_t* out, int nx, int nz, data_t dx, data_t dz, data_t dt, const data_t * par0,  const data_t * par1, const data_t * par2, data_t a0, data_t a1, data_t a2)
{
    int ix = threadIdx.x + blockIdx.x*BLOCK_SIZE;

    data_t coef[2] = {2.0/3,-1.0/12};
    data_t scoef[4] = {11.0/6, -3, 1.5, -1.0/3};
    data_t h0 = 17.0/48;
    int nc1=4, nc2=6;
    data_t sumx=0, sumz=0;

    // SAT = - H-1 (-f1.Dx.in1 + f2.Sz.in2 -f3.in3/dt)_0
    // f3 is often P or S impedance (rho*Vp or rho*Vs) divided by 2 ; f3=par[2] is a boundary array only

    // top left
    if (ix<nc1)
    {
        // (Dx.in1)_0
        sumx=0;
        for (int j=0; j<nc2; j++){
            sumx += bnd_coef1[ix*nc2+j] * in0[j*nz];
        }

        // (Sz.in2)_0
        sumz = 0;
        for (int iz = 0; iz < 4; iz++){
            sumz += scoef[iz] * in1[ix*nz+iz];
        }
        out[ix*nz] = add*out[ix*nz] - 1.0 /(dz * h0) * (-a0*par0[ix*nz]*sumx/dx + a1*par1[ix*nz]*sumz/dz - a2*par2[ix]*in2[ix*nz]/dt);
    }

    // top middle
    else if (ix<nx-nc1)
    {
        // (Dx.in1)_0
        sumx=0;
        for (int j=1; j<=2; j++){
            sumx += coef[j-1] * (in0[(ix+j)*nz]-in0[(ix-j)*nz]);
        }

        // (Sz.in2)_0
        sumz = 0;
        for (int iz = 0; iz < 4; iz++){
            sumz += scoef[iz] * in1[ix*nz+iz];
        }

        out[ix*nz] = add*out[ix*nz] - 1.0 /(dz * h0) * (-a0*par0[ix*nz]*sumx/dx + a1*par1[ix*nz]*sumz/dz - a2*par2[ix]*in2[ix*nz]/dt);
    }

    // top right
    else if (ix<nx)
    {
        int jx=nx-ix-1;
        // (Dx.in1)_0
        sumx=0;
        for (int j=0; j<nc2; j++){
            sumx -= bnd_coef1[jx*nc2+j] * in0[(nx-1-j)*nz];
        }

        // (Sz.in2)_0
        sumz = 0;
        for (int iz = 0; iz < 4; iz++){
            sumz += scoef[iz] * in1[(nx-1-jx)*nz+iz];
        }
        out[(nx-1-jx)*nz] = add*out[(nx-1-jx)*nz] - 1.0 /(dz * h0) * (-a0*par0[(nx-1-jx)*nz]*sumx/dx + a1*par1[(nx-1-jx)*nz]*sumz/dz  - a2*par2[(nx-1-jx)]*in2[(nx-1-jx)*nz]/dt);
    }
}

__global__ void cudaEsatAbsorbingBottom(bool add, const data_t* in0, const data_t* in1, const data_t* in2, data_t* out, int nx, int nz, data_t dx, data_t dz, data_t dt, const data_t * par0,  const data_t * par1, const data_t * par2, data_t a0, data_t a1, data_t a2)
{
    int ix = threadIdx.x + blockIdx.x*BLOCK_SIZE;

    data_t coef[2] = {2.0/3,-1.0/12};
    data_t scoef[4] = {11.0/6, -3, 1.5, -1.0/3};
    data_t h0 = 17.0/48;
    int nc1=4, nc2=6;
    data_t sumx=0, sumz=0;

    // SAT = - H-1 (f1.Dx.in1 + f2.Sz.in2 - f3.in3/dt)_0

    // bottom left
    if (ix<nc1)
    {
        // (Dx.in1)_0
        sumx=0;
        for (int j=0; j<nc2; j++){
            sumx += bnd_coef1[ix*nc2+j] * in0[j*nz+nz-1];
        }

        // (Sz.in2)_0
        sumz = 0;
        for (int iz = 0; iz < 4; iz++){
            sumz += scoef[iz] * in1[ix*nz+nz-1-iz];
        }
        out[ix*nz+nz-1] = add*out[ix*nz+nz-1] - 1.0 /(dz * h0) * (a0*par0[ix*nz+nz-1]*sumx/dx + a1*par1[ix*nz+nz-1]*sumz/dz - a2*par2[ix]*in2[ix*nz+nz-1]/dt);
    }

    // bottom middle
    else if (ix<nx-nc1)
    {
        // (Dx.in1)_0
        sumx=0;
        for (int j=1; j<=2; j++){
            sumx += coef[j-1] * (in0[(ix+j)*nz+nz-1]-in0[(ix-j)*nz+nz-1]);
        }

        // (Sz.in2)_0
        sumz = 0;
        for (int iz = 0; iz < 4; iz++){
            sumz += scoef[iz] * in1[ix*nz+nz-1-iz];
        }

        out[ix*nz+nz-1] = add*out[ix*nz+nz-1] - 1.0 /(dz * h0) * (a0*par0[ix*nz+nz-1]*sumx/dx + a1*par1[ix*nz+nz-1]*sumz/dz - a2*par2[ix]*in2[ix*nz+nz-1]/dt);
    }

    // bottom right
    else if (ix<nx)
    {
        int jx=nx-ix-1;
        // (Dx.in1)_0
        sumx=0;
        for (int j=0; j<nc2; j++){
            sumx -= bnd_coef1[jx*nc2+j] * in0[(nx-1-j)*nz+nz-1];
        }

        // (Sz.in2)_0
        sumz = 0;
        for (int iz = 0; iz < 4; iz++){
            sumz += scoef[iz] * in1[(nx-1-jx)*nz+nz-1-iz];
        }
        out[(nx-1-jx)*nz+nz-1] = add*out[(nx-1-jx)*nz+nz-1] - 1.0 /(dz * h0) * (a0*par0[(nx-1-jx)*nz+nz-1]*sumx/dx + a1*par1[(nx-1-jx)*nz+nz-1]*sumz/dz - a2*par2[nx-1-jx]*in2[(nx-1-jx)*nz+nz-1]/dt);
    }
}

__global__ void cudaEsatAbsorbingLeft(bool add, const data_t* in0, const data_t* in1, const data_t* in2, data_t* out, int nx, int nz, data_t dx, data_t dz, data_t dt, const data_t * par0,  const data_t * par1, const data_t * par2, data_t a0, data_t a1, data_t a2)
{
    int iz = threadIdx.y + blockIdx.y*BLOCK_SIZE;

    data_t coef[2] = {2.0/3,-1.0/12};
    data_t scoef[4] = {11.0/6, -3, 1.5, -1.0/3};
    data_t h0 = 17.0/48;
    int nc1=4, nc2=6;
    data_t sumx=0, sumz=0;

    // SAT = - H-1 (-f1.Dz.in1 + f2.Sx.in2 - f3.in3/dt)_0

    // left top
    if (iz<nc1)
    {
        // (Dz.in1)_0
        sumz=0;
        for (int j=0; j<nc2; j++){
            sumz += bnd_coef1[iz*nc2+j] * in0[j];
        }

        // (Sx.in2)_0
        sumx = 0;
        for (int ix = 0; ix < 4; ix++){
            sumx += scoef[ix] * in1[ix*nz+iz];
        }
        out[iz] = add*out[iz] - 1.0 /(dx * h0) * (-a0*par0[iz]*sumz/dz + a1*par1[iz]*sumx/dx - a2*par2[iz]*in2[iz]/dt);
    }

    // left middle
    else if (iz<nz-nc1)
    {
        // (Dz.in1)_0
        sumz=0;
        for (int j=1; j<=2; j++){
            sumz += coef[j-1] * (in0[iz+j]-in0[iz-j]);
        }

        // (Sx.in2)_0
        sumx = 0;
        for (int ix = 0; ix < 4; ix++){
            sumx += scoef[ix] * in1[ix*nz+iz];
        }

        out[iz] = add*out[iz] - 1.0 /(dx * h0) * (-a0*par0[iz]*sumz/dz + a1*par1[iz]*sumx/dx - a2*par2[iz]*in2[iz]/dt);
    }

    // left bottom
    else if (iz<nz)
    {
        int jz=nz-iz-1;
        // (Dz.in1)_0
        sumz=0;
        for (int j=0; j<nc2; j++){
            sumz -= bnd_coef1[jz*nc2+j] * in0[nz-1-j];
        }

        // (Sx.in2)_0
        sumx = 0;
        for (int ix = 0; ix < 4; ix++){
            sumx += scoef[ix] * in1[ix*nz+nz-1-jz];
        }
        out[nz-1-jz] = add*out[nz-1-jz] - 1.0 /(dx * h0) * (-a0*par0[nz-1-jz]*sumz/dz + a1*par1[nz-1-jz]*sumx/dx - a2*par2[nz-1-jz]*in2[nz-1-jz]/dt);
    }
}

__global__ void cudaEsatAbsorbingRight(bool add, const data_t* in0, const data_t* in1, const data_t* in2, data_t* out, int nx, int nz, data_t dx, data_t dz, data_t dt, const data_t * par0,  const data_t * par1, const data_t * par2, data_t a0, data_t a1, data_t a2)
{
    int iz = threadIdx.y + blockIdx.y*BLOCK_SIZE;

    data_t coef[2] = {2.0/3,-1.0/12};
    data_t scoef[4] = {11.0/6, -3, 1.5, -1.0/3};
    data_t h0 = 17.0/48;
    int nc1=4, nc2=6;
    data_t sumx=0, sumz=0;

    // SAT = - H-1 (f1.Dz.in1 + f2.Sx.in2 - f3.in3/dt)_0

    // right top
    if (iz<nc1)
    {
        // (Dz.in1)_0
        sumz=0;
        for (int j=0; j<nc2; j++){
            sumz += bnd_coef1[iz*nc2+j] * in0[(nx-1)*nz+j];
        }

        // (Sx.in2)_0
        sumx = 0;
        for (int ix = 0; ix < 4; ix++){
            sumx += scoef[ix] * in1[(nx-1-ix)*nz+iz];
        }
        out[(nx-1)*nz+iz] = add*out[(nx-1)*nz+iz] - 1.0 /(dx * h0) * (a0*par0[(nx-1)*nz+iz]*sumz/dz + a1*par1[(nx-1)*nz+iz]*sumx/dx - a2*par2[iz]*in2[(nx-1)*nz+iz]/dt);
    }

    // right middle
    else if (iz<nz-nc1)
    {
        // (Dz.in1)_0
        sumz=0;
        for (int j=1; j<=2; j++){
            sumz += coef[j-1] * (in0[(nx-1)*nz+iz+j]-in0[(nx-1)*nz+iz-j]);
        }

        // (Sx.in2)_0
        sumx = 0;
        for (int ix = 0; ix < 4; ix++){
            sumx += scoef[ix] * in1[(nx-1-ix)*nz+iz];
        }

        out[(nx-1)*nz+iz] = add*out[(nx-1)*nz+iz] - 1.0 /(dx * h0) * (a0*par0[(nx-1)*nz+iz]*sumz/dz + a1*par1[(nx-1)*nz+iz]*sumx/dx - a2*par2[iz]*in2[(nx-1)*nz+iz]/dt);
    }

    // right bottom
    else if (iz<nz)
    {
        int jz=nz-1-iz;
        // (Dz.in1)_0
        sumz=0;
        for (int j=0; j<nc2; j++){
            sumz -= bnd_coef1[jz*nc2+j] * in0[(nx-1)*nz+nz-1-j];
        }

        // (Sx.in2)_0
        sumx = 0;
        for (int ix = 0; ix < 4; ix++){
            sumx += scoef[ix] * in1[(nx-1-ix)*nz+nz-1-jz];
        }
        out[(nx-1)*nz+nz-1-jz] = add*out[(nx-1)*nz+nz-1-jz] - 1.0 /(dx * h0) * (a0*par0[(nx-1)*nz+nz-1-jz]*sumz/dz + a1*par1[(nx-1)*nz+nz-1-jz]*sumx/dx - a2*par2[nz-1-jz]*in2[(nx-1)*nz+nz-1-jz]/dt);
    }
}

__global__ void cudaDzTop(bool add, const data_t * in, data_t * out, int nx, int nz, data_t d, const data_t * par, data_t a){
    
    int ix = threadIdx.x + blockIdx.x*BLOCK_SIZE;

    data_t h0 = 17.0/48;
    int nc2=6;
    data_t sc = a /(d*d*h0);
    data_t sum=0;

    // additional SAT = - H-1 (-f.Dz.in)_0
    if (ix<nx)
    {
        sum=0;
        for (int j=0; j<nc2; j++){
            sum -= bnd_coef1[j] * in[ix*nz+j];
        }
        out[ix*nz] = add*out[ix*nz] - sc * par[ix*nz]*sum;
    }
}
__global__ void cudaDzBottom(bool add, const data_t * in, data_t * out, int nx, int nz, data_t d, const data_t * par, data_t a){
    
    int ix = threadIdx.x + blockIdx.x*BLOCK_SIZE;

    data_t h0 = 17.0/48;
    int nc2=6;
    data_t sc = a /(d*d*h0);
    data_t sum=0;

    // additional SAT = - H-1 (f.Dz.in)_0
    if (ix<nx)
    {
        sum=0;
        for (int j=0; j<nc2; j++){
            sum -= bnd_coef1[j] * in[ix*nz+nz-1-j];
        }
        out[ix*nz+nz-1] = add*out[ix*nz+nz-1] - sc * par[ix*nz+nz-1]*sum;
    }
}
__global__ void cudaDxLeft(bool add, const data_t * in, data_t * out, int nx, int nz, data_t d, const data_t * par, data_t a){
    
    int iz = threadIdx.y + blockIdx.y*BLOCK_SIZE;

    data_t h0 = 17.0/48;
    int nc2=6;
    data_t sc = a /(d*d*h0);
    data_t sum=0;

    // additional SAT = - H-1 (-f.Dx.in)_0
    if (iz<nz)
    {
        sum=0;
        for (int j=0; j<nc2; j++){
            sum -= bnd_coef1[j] * in[j*nz+iz];
        }
        out[iz] = add*out[iz] - sc * par[iz]*sum;
    }
}
__global__ void cudaDxRight(bool add, const data_t * in, data_t * out, int nx, int nz, data_t d, const data_t * par, data_t a){
    
    int iz = threadIdx.y + blockIdx.y*BLOCK_SIZE;

    data_t h0 = 17.0/48;
    int nc2=6;
    data_t sc = a /(d*d*h0);
    data_t sum=0;

    // additional SAT = - H-1 (f.Dx.in)_0
    if (iz<nz)
    {
        sum=0;
        for (int j=0; j<nc2; j++){
            sum -= bnd_coef1[j] * in[(nx-1-j)*nz+iz];
        }
        out[(nx-1)*nz+iz] = add*out[(nx-1)*nz+iz] - sc * par[(nx-1)*nz+iz]*sum;
    }
}

__global__ void cudaScaleTopBottom(data_t* in, int nx, int nz, data_t dx, data_t dz, const data_t* par, data_t dt, bool top, bool bottom){

    int ix = threadIdx.x + blockIdx.x*BLOCK_SIZE + 1;

    data_t h0 = 17.0/48;
    data_t scx=0, scz=0;
    const data_t * par0 = par, *par1=par+nx*nz, *par2=par+2*nx*nz;
    data_t * in0 = in, *in1=in+nx*nz;

    if (blockIdx.y==0) // top
    {
        if (ix<nx-1)
        {
            scx = top*sqrt(par1[ix*nz]/par2[ix*nz])*dt / (2 * dz * h0);
            scz = scx*sqrt((par0[ix*nz]+2*par1[ix*nz])/par1[ix*nz]);
            in0[ix*nz] /= (1+scx);
            in1[ix*nz] /= (1+scz);
        }
    }
    else // bottom
    {
        if (ix<nx-1)
        {
            scx = bottom*sqrt(par1[ix*nz+nz-1]/par2[ix*nz+nz-1])*dt / (2 * dz * h0);
            scz = scx*sqrt((par0[ix*nz+nz-1]+2*par1[ix*nz+nz-1])/par1[ix*nz+nz-1]);
            in0[ix*nz+nz-1] /= (1+scx);
            in1[ix*nz+nz-1] /= (1+scz);
        }
    }
}

__global__ void cudaScaleLeftRight(data_t* in, int nx, int nz, data_t dx, data_t dz, const data_t* par, data_t dt, bool top, bool bottom, bool left, bool right){

    int iz = threadIdx.y + blockIdx.y*BLOCK_SIZE;

    data_t h0 = 17.0/48;
    data_t scx=0, scz=0;
    const data_t * par0 = par, *par1=par+nx*nz, *par2=par+2*nx*nz;
    data_t * in0 = in, *in1=in+nx*nz;

    if (blockIdx.x==0) // left
    {
        if (iz<nz)
        {
            scx = left*sqrt((par0[iz]+2*par1[iz])/par2[iz])*dt / (2 * dx * h0);
            scz = scx*sqrt(par1[iz]/(par0[iz]+2*par1[iz]));
            if (iz==0) { // left top
                scx += top*sqrt(par1[0]/par2[0])*dt / (2  * dz * h0);
                scz += top*sqrt((par0[0]+2*par1[0])/par2[0])*dt / (2  * dz * h0);
            }
            else if (iz==nz-1) { // left bottom
                scx += bottom*sqrt(par1[nz-1]/par2[nz-1])*dt / (2 * dz * h0);
                scz += bottom*sqrt((par0[nz-1]+2*par1[nz-1])/par2[nz-1])*dt / (2 * dz * h0);
            }
            in0[iz] = in0[iz] / (1+scx);
            in1[iz] = in1[iz] / (1+scz);
        }
    }
    else // right
    {
        if (iz<nz)
        {
            scx = right*sqrt((par0[(nx-1)*nz+iz]+2*par1[(nx-1)*nz+iz])/par2[(nx-1)*nz+iz])*dt / (2 * dx * h0);
            scz = scx*sqrt(par1[(nx-1)*nz+iz]/(par0[(nx-1)*nz+iz]+2*par1[(nx-1)*nz+iz]));
            if (iz==0) { // right top
                scx += top*sqrt(par1[(nx-1)*nz]/par2[(nx-1)*nz])*dt / (2 * dz * h0);
                scz += top*sqrt((par0[(nx-1)*nz]+2*par1[(nx-1)*nz])/par2[(nx-1)*nz])*dt / (2 * dz * h0);
            }
            else if (iz==nz-1) { // right bottom
                scx += bottom*sqrt(par1[(nx-1)*nz+nz-1]/par2[(nx-1)*nz+nz-1])*dt / (2 * dz * h0);
                scz += bottom*sqrt((par0[(nx-1)*nz+nz-1]+2*par1[(nx-1)*nz+nz-1])/par2[(nx-1)*nz+nz-1])*dt / (2 * dz * h0);
            }
            in0[(nx-1)*nz+iz] = in0[(nx-1)*nz+iz] / (1+scx);
            in1[(nx-1)*nz+iz] = in1[(nx-1)*nz+iz] / (1+scz);
        }
    }
}

__global__ void cudaScaleLeftRightVTI(data_t* in, int nx, int nz, data_t dx, data_t dz, const data_t* par, data_t dt, bool top, bool bottom, bool left, bool right){

    int iz = threadIdx.y + blockIdx.y*BLOCK_SIZE;

    data_t h0 = 17.0/48;
    data_t scx=0, scz=0;
    const data_t * par0 = par, *par1=par+nx*nz, *par2=par+2*nx*nz, *par4=par+4*nx*nz;
    data_t * in0 = in, *in1=in+nx*nz;

    if (blockIdx.x==0) // left
    {
        if (iz<nz)
        {
            scx = left*sqrt((1+2*par4[iz])*(par0[iz]+2*par1[iz])/par2[iz])*dt / (2 * dx * h0);
            scz = left*sqrt(par1[iz]/par2[iz])*dt / (2 * dx * h0);
            if (iz==0) { // left top
                scx += top*sqrt(par1[0]/par2[0])*dt / (2  * dz * h0);
                scz += top*sqrt((par0[0]+2*par1[0])/par2[0])*dt / (2  * dz * h0);
            }
            else if (iz==nz-1) { // left bottom
                scx += bottom*sqrt(par1[nz-1]/par2[nz-1])*dt / (2 * dz * h0);
                scz += bottom*sqrt((par0[nz-1]+2*par1[nz-1])/par2[nz-1])*dt / (2 * dz * h0);
            }
            in0[iz] = in0[iz] / (1+scx);
            in1[iz] = in1[iz] / (1+scz);
        }
    }
    else // right
    {
        if (iz<nz)
        {
            scx = right*sqrt((1+2*par4[(nx-1)*nz+iz])*(par0[(nx-1)*nz+iz]+2*par1[(nx-1)*nz+iz])/par2[(nx-1)*nz+iz])*dt / (2 * dx * h0);
            scz = right*sqrt(par1[(nx-1)*nz+iz]/par2[(nx-1)*nz+iz])*dt / (2 * dx * h0);
            if (iz==0) { // right top
                scx += top*sqrt(par1[(nx-1)*nz]/par2[(nx-1)*nz])*dt / (2 * dz * h0);
                scz += top*sqrt((par0[(nx-1)*nz]+2*par1[(nx-1)*nz])/par2[(nx-1)*nz])*dt / (2 * dz * h0);
            }
            else if (iz==nz-1) { // right bottom
                scx += bottom*sqrt(par1[(nx-1)*nz+nz-1]/par2[(nx-1)*nz+nz-1])*dt / (2 * dz * h0);
                scz += bottom*sqrt((par0[(nx-1)*nz+nz-1]+2*par1[(nx-1)*nz+nz-1])/par2[(nx-1)*nz+nz-1])*dt / (2 * dz * h0);
            }
            in0[(nx-1)*nz+iz] = in0[(nx-1)*nz+iz] / (1+scx);
            in1[(nx-1)*nz+iz] = in1[(nx-1)*nz+iz] / (1+scz);
        }
    }
}

__global__ void cudaTaperTop(data_t* in, int nx, int nz, int taper, data_t a){

    int ix = threadIdx.x + blockIdx.x*blockDim.x;
    int iz = threadIdx.y;

    data_t val = cos(a*0.5*M_PI*(taper-1-iz)/taper);
    if (ix<nx) in[ix*nz+iz] *= val*val;
}
__global__ void cudaTaperBottom(data_t* in, int nx, int nz, int taper, data_t a){

    int ix = threadIdx.x + blockIdx.x*blockDim.x;
    int iz = threadIdx.y;

    data_t val = cos(a*0.5*M_PI*(taper-1-iz)/taper);
    if (ix<nx) in[ix*nz+nz-1-iz] *= val*val;
}
__global__ void cudaTaperLeft(data_t* in, int nx, int nz, int taper, data_t a){

    int iz = threadIdx.y + blockIdx.y*blockDim.y;
    int ix = threadIdx.x;

    data_t val = cos(a*0.5*M_PI*(taper-1-ix)/taper);
    if (iz<nz) in[ix*nz+iz] *= val*val;
}
__global__ void cudaTaperRight(data_t* in, int nx, int nz, int taper, data_t a){

    int iz = threadIdx.y + blockIdx.y*blockDim.y;
    int ix = threadIdx.x;

    data_t val = cos(a*0.5*M_PI*(taper-1-ix)/taper);
    if (iz<nz) in[(nx-1-ix)*nz+iz] *= val*val;
}

__global__ void cudaTimeStep(const data_t * prev, const data_t * curr, data_t * next, const data_t * par, int nx, int nz, data_t dt){

    int ix = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    int iz = threadIdx.y + blockIdx.y*BLOCK_SIZE;

    if (ix<nx && iz<nz) next[ix*nz+iz] = dt*dt*next[ix*nz+iz]/par[ix*nz+iz] + 2*curr[ix*nz+iz] - prev[ix*nz+iz];
}

__global__ void cudaInjectDM3(const data_t ** in, data_t * out, int nx, int nz, int nt, int ntr, int it, int itr_min, int itr_max, const int * xind, const int * zind, const data_t * xw, const data_t * zw)
{
    int ix = threadIdx.x;
    int iz = threadIdx.y;
    int itr = blockIdx.x+itr_min;

    const int (* p_xind) [3] = (const int (*) [3]) xind;
    const int (* p_zind) [3] = (const int (*) [3]) zind;
    const data_t (* p_xw) [2][3] = (const data_t (*) [2][3]) xw;
    const data_t (* p_zw) [2][3] = (const data_t (*) [2][3]) zw;

    data_t val = p_xw[itr][0][ix] * p_zw[itr][0][iz] * in[0][itr*nt+it];
    atomicAdd(out + p_xind[itr][ix]*nz+p_zind[itr][iz], val);
}

__global__ void cudaExtractDM3(const data_t * in, data_t ** out, int nx, int nz, int nt, int ntr, int it, int itr_min, int itr_max, const int * xind, const int * zind, const data_t * xw, const data_t * zw)
{
    int ix = threadIdx.x;
    int iz = threadIdx.y;
    int itr = blockIdx.x+itr_min;

    const int (* p_xind) [3] = (const int (*) [3]) xind;
    const int (* p_zind) [3] = (const int (*) [3]) zind;
    const data_t (* p_xw) [2][3] = (const data_t (*) [2][3]) xw;
    const data_t (* p_zw) [2][3] = (const data_t (*) [2][3]) zw;

    data_t val = p_xw[itr][1][ix] * p_zw[itr][1][iz] * in[p_xind[itr][ix]*nz+p_zind[itr][iz]];
    atomicAdd(out[0]+itr*nt+it, val);
}

__global__ void cudaInjectDDM3(const data_t ** in, data_t * out, int nx, int nz, int nt, int ntr, int it, int itr_min, int itr_max, const int * xind, const int * zind, const data_t * xw, const data_t * zw)
{
    int ix = threadIdx.x;
    int iz = threadIdx.y;
    int itr = blockIdx.x+itr_min;

    const int (* p_xind) [3] = (const int (*) [3]) xind;
    const int (* p_zind) [3] = (const int (*) [3]) zind;
    const data_t (* p_xw) [2][6] = (const data_t (*) [2][6]) xw;
    const data_t (* p_zw) [2][6] = (const data_t (*) [2][6]) zw;

    data_t val = (-p_xw[itr][0][ix+3] * p_zw[itr][0][iz] * in[0][itr*nt+it] - p_xw[itr][0][ix] * p_zw[itr][0][iz+3] * in[1][itr*nt+it]);
    atomicAdd(out + p_xind[itr][ix]*nz+p_zind[itr][iz], val);
}

__global__ void cudaExtractDDM3(const data_t * in, data_t ** out, int nx, int nz, int nt, int ntr, int it, int itr_min, int itr_max, const int * xind, const int * zind, const data_t * xw, const data_t * zw)
{
    int ix = threadIdx.x;
    int iz = threadIdx.y;
    int itr = blockIdx.x+itr_min;

    const int (* p_xind) [3] = (const int (*) [3]) xind;
    const int (* p_zind) [3] = (const int (*) [3]) zind;
    const data_t (* p_xw) [2][6] = (const data_t (*) [2][6]) xw;
    const data_t (* p_zw) [2][6] = (const data_t (*) [2][6]) zw;

    data_t val = - p_xw[itr][1][ix+3] * p_zw[itr][1][iz] * in[p_xind[itr][ix]*nz+p_zind[itr][iz]];
    atomicAdd(out[0]+itr*nt+it, val);
    val = - p_xw[itr][1][ix] * p_zw[itr][1][iz+3] * in[p_xind[itr][ix]*nz+p_zind[itr][iz]];
    atomicAdd(out[1]+itr*nt+it, val);
}

__global__ void cudaInjectDIPM3(const data_t ** in, data_t * out, int nx, int nz, int nt, int ntr, int it, int itr_min, int itr_max, const int * xind, const int * zind, const data_t * xw, const data_t * zw)
{
    int ix = threadIdx.x;
    int iz = threadIdx.y;
    int itr = blockIdx.x+itr_min;

    const int (* p_xind) [6] = (const int (*) [6]) xind;
    const int (* p_zind) [6] = (const int (*) [6]) zind;
    const data_t (* p_xw) [2][6] = (const data_t (*) [2][6]) xw;
    const data_t (* p_zw) [2][6] = (const data_t (*) [2][6]) zw;

    data_t val = -p_xw[itr][0][ix] * p_zw[itr][0][iz] * in[0][itr*nt+it];
    atomicAdd(out + p_xind[itr][ix]*nz+p_zind[itr][iz], val);
    val = p_xw[itr][0][ix+3] * p_zw[itr][0][iz+3] * in[0][itr*nt+it];
    atomicAdd(out + p_xind[itr][ix+3]*nz+p_zind[itr][iz+3], val);
}

__global__ void cudaExtractDIPM3(const data_t * in, data_t ** out, int nx, int nz, int nt, int ntr, int it, int itr_min, int itr_max, const int * xind, const int * zind, const data_t * xw, const data_t * zw)
{
    int ix = threadIdx.x;
    int iz = threadIdx.y;
    int itr = blockIdx.x+itr_min;

    const int (* p_xind) [6] = (const int (*) [6]) xind;
    const int (* p_zind) [6] = (const int (*) [6]) zind;
    const data_t (* p_xw) [2][6] = (const data_t (*) [2][6]) xw;
    const data_t (* p_zw) [2][6] = (const data_t (*) [2][6]) zw;

    data_t val = p_xw[itr][1][ix+3] * p_zw[itr][1][iz+3] * in[p_xind[itr][ix+3]*nz+p_zind[itr][iz+3]]
    - p_xw[itr][1][ix] * p_zw[itr][1][iz] * in[p_xind[itr][ix]*nz+p_zind[itr][iz]];
    atomicAdd(out[0]+itr*nt+it, val);
}

__global__ void cudaComputeGradients(const data_t * model, const data_t * u_for, const data_t * curr, const data_t * u_x, const data_t * u_z, data_t * tmp, data_t * grad, int nx, int nz, int nt, int sub, int it, data_t dx, data_t dz, data_t dt){

    int ix = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    int iz = threadIdx.y + blockIdx.y*BLOCK_SIZE;

    int nxz=nx*nz;
    const data_t * pfor0x=u_for;
    const data_t * pfor0z=u_for+nxz;
    const data_t * pfor1x=u_for+2*nxz;
    const data_t * pfor1z=u_for+3*nxz;
    const data_t * pfor2x=u_for+4*nxz;
    const data_t * pfor2z=u_for+5*nxz;
    const data_t *padjx = curr, *padjz=curr+nxz;
    const data_t *padjx_x = u_x, *padjz_x=u_x+nxz;
    const data_t *padjx_z = u_z, *padjz_z=u_z+nxz;
    data_t *pforx_x=tmp, *pforz_z=tmp+nxz, *pforz_x=tmp+2*nxz, *pforx_z=tmp+3*nxz;
    data_t *gla = grad, *gmu=grad+nxz, *grho=grad+2*nxz;

    int i=0;
    if (ix<nx && iz<nz)
    {
        i=ix*nz+iz;
        if (it>0 && it<nt/sub){
            gla[i] += dt*(padjx_x[i] + padjz_z[i])*(pforx_x[i] + pforz_z[i]);
            gmu[i] += dt*((padjx_z[i] + padjz_x[i])*(pforz_x[i] + pforx_z[i]) + 2*padjx_x[i]*pforx_x[i] + 2*padjz_z[i]*pforz_z[i]);
            grho[i] += 1.0/dt*(padjx[i]*(pfor2x[i]-2*pfor1x[i]+pfor0x[i]) + padjz[i]*(pfor2z[i]-2*pfor1z[i]+pfor0z[i]));
        }
        else if (it==0){
            gla[i] += 0.5*dt*(padjx_x[i] + padjz_z[i])*(pforx_x[i] + pforz_z[i]);
            gmu[i] += 0.5*dt*((padjx_z[i] + padjz_x[i])*(pforz_x[i] + pforx_z[i]) + 2*padjx_x[i]*pforx_x[i] + 2*padjz_z[i]*pforz_z[i]);
            grho[i] += 1.0/dt*(padjx[i]*(pfor2x[i]-pfor1x[i]) + padjz[i]*(pfor2z[i]-pfor1z[i]));
        }
        else{
            gla[i] += dt*(padjx_x[i] + padjz_z[i])*(pforx_x[i] + pforz_z[i]);
            gmu[i] += dt*((padjx_z[i] + padjz_x[i])*(pforz_x[i] + pforx_z[i]) + 2*padjx_x[i]*pforx_x[i] + 2*padjz_z[i]*pforz_z[i]);
            grho[i] += 1.0/dt*(padjx[i]*(-pfor1x[i]+pfor0x[i]) + padjz[i]*(-pfor1z[i]+pfor0z[i]));
        }
    }
}

__global__ void cudaComputeGradientsVTI(const data_t * model, const data_t * u_for, const data_t * curr, const data_t * u_x, const data_t * u_z, data_t * tmp, data_t * grad, int nx, int nz, int nt, int sub, int it, data_t dx, data_t dz, data_t dt){

    int ix = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    int iz = threadIdx.y + blockIdx.y*BLOCK_SIZE;

    int nxz=nx*nz;
    const data_t * pfor0x=u_for;
    const data_t * pfor0z=u_for+nxz;
    const data_t * pfor1x=u_for+2*nxz;
    const data_t * pfor1z=u_for+3*nxz;
    const data_t * pfor2x=u_for+4*nxz;
    const data_t * pfor2z=u_for+5*nxz;
    const data_t *padjx = curr, *padjz=curr+nxz;
    const data_t *padjx_x = u_x, *padjz_x=u_x+nxz;
    const data_t *padjx_z = u_z, *padjz_z=u_z+nxz;
    data_t *pforx_x=tmp, *pforz_z=tmp+nxz, *pforz_x=tmp+2*nxz, *pforx_z=tmp+3*nxz;
    data_t *gla = grad, *gmu=grad+nxz, *grho=grad+2*nxz, *gdel=grad+3*nxz, *geps=grad+4*nxz;
    const data_t *pm0 = model, *pm1=model+nxz, *pm3=model+3*nxz, *pm4=model+4*nxz;

    data_t val1=0, val2=0, val3=0, val4=0, del=0;
    int i=0;
    if (ix<nx && iz<nz)
    {
        i=ix*nz+iz;
        del = ((pm3[i]+pm1[i])*(pm3[i]+pm1[i]) - (pm0[i]+pm1[i])*(pm0[i]+pm1[i])) / (2*(pm0[i]+pm1[i])*(pm0[i]+2*pm1[i]));
        val1 = sqrt(2*(pm0[i]+2*pm1[i])*(pm0[i]+pm1[i])*del + (pm0[i]+pm1[i])*(pm0[i]+pm1[i]));
        val2 = ((1+2*del)*pm0[i] + (1+3*del)*pm1[i])/val1; // d(C13)/d(lambda)
        val3 = ((1+3*del)*pm0[i] + (1+4*del)*pm1[i])/val1 - 1; // d(C13)/d(mu)
        val4 = (pm0[i]+2*pm1[i])*(pm0[i]+pm1[i])/val1; // d(C13)/d(delta)
        if (it>0 && it<nt/sub){
            gla[i] += dt*((1+2*pm4[i])*padjx_x[i]*pforx_x[i] + padjz_z[i]*pforz_z[i] + val2*(padjx_x[i]*pforz_z[i] + padjz_z[i]*pforx_x[i])); // lambda gradient
            gmu[i] += dt*((padjx_z[i] + padjz_x[i])*(pforz_x[i] + pforx_z[i]) + 2*(1+2*pm4[i])*padjx_x[i]*pforx_x[i] + 2*padjz_z[i]*pforz_z[i] + val3*(padjx_x[i]*pforz_z[i] + padjz_z[i]*pforx_x[i])); // mu gradient
            grho[i] += 1.0/dt*(padjx[i]*(pfor2x[i]-2*pfor1x[i]+pfor0x[i]) + padjz[i]*(pfor2z[i]-2*pfor1z[i]+pfor0z[i])); // rho gradient
            gdel[i] += dt*val4*2*(padjx_x[i]*pforx_x[i] + padjz_z[i]*pforz_z[i]); // delta gradient
            geps[i] += dt*2*(pm0[i]+2*pm1[i])*padjx_x[i]*pforx_x[i]; // epsilon gradient
        }
        else if (it==0) {
            gla[i] += 0.5*dt*((1+2*pm4[i])*padjx_x[i]*pforx_x[i] + padjz_z[i]*pforz_z[i] + val2*(padjx_x[i]*pforz_z[i] + padjz_z[i]*pforx_x[i])); // lambda gradient
            gmu[i] += 0.5*dt*((padjx_z[i] + padjz_x[i])*(pforz_x[i] + pforx_z[i]) + 2*(1+2*pm4[i])*padjx_x[i]*pforx_x[i] + 2*padjz_z[i]*pforz_z[i] + val3*(padjx_x[i]*pforz_z[i] + padjz_z[i]*pforx_x[i])); // mu gradient
            grho[i] += 1.0/dt*(padjx[i]*(pfor2x[i]-pfor1x[i]) + padjz[i]*(pfor2z[i]-pfor1z[i])); // rho gradient
            gdel[i] += 0.5*dt*val4*2*(padjx_x[i]*pforx_x[i] + padjz_z[i]*pforz_z[i]); // delta gradient
            geps[i] += 0.5*dt*2*(pm0[i]+2*pm1[i])*padjx_x[i]*pforx_x[i]; // epsilon gradient
        }
        else{
            gla[i] += dt*((1+2*pm4[i])*padjx_x[i]*pforx_x[i] + padjz_z[i]*pforz_z[i] + val2*(padjx_x[i]*pforz_z[i] + padjz_z[i]*pforx_x[i])); // lambda gradient
            gmu[i] += dt*((padjx_z[i] + padjz_x[i])*(pforz_x[i] + pforx_z[i]) + 2*(1+2*pm4[i])*padjx_x[i]*pforx_x[i] + 2*padjz_z[i]*pforz_z[i] + val3*(padjx_x[i]*pforz_z[i] + padjz_z[i]*pforx_x[i])); // mu gradient
            grho[i] += 1.0/dt*(padjx[i]*(-pfor1x[i]+pfor0x[i]) + padjz[i]*(-pfor1z[i]+pfor0z[i])); // rho gradient
            gdel[i] += dt*val4*2*(padjx_x[i]*pforx_x[i] + padjz_z[i]*pforz_z[i]); // delta gradient
            geps[i] += dt*2*(pm0[i]+2*pm1[i])*padjx_x[i]*pforx_x[i]; // epsilon gradient
        }
    }
}

// ################################ C++ wrappers ##################################

hipStream_t streams[5];

void Dz_gpu(bool add, const data_t* in, data_t* out, int nx, int nz, data_t d, int stream1, int stream2){
    
    dim3 threads1(BLOCK_SIZE,BLOCK_SIZE);
    dim3 blocks1((nx+BLOCK_SIZE-1)/BLOCK_SIZE,(nz-8+BLOCK_SIZE-1)/BLOCK_SIZE);
    dim3 threads2(BLOCK_SIZE,4);
    dim3 blocks2((nx+BLOCK_SIZE-1)/BLOCK_SIZE,2);

    cudaDz_interior<<<blocks1,threads1,0,streams[stream1]>>>(add, in, out, nx, nz, d);
    cudaKernelError();
    cudaDz_bnd<<<blocks2,threads2,0,streams[stream2]>>>(add, in, out, nx, nz, d);
    cudaKernelError();
}

void Dx_gpu(bool add, const data_t* in, data_t* out, int nx, int nz, data_t d, int stream1, int stream2){

    dim3 threads1(BLOCK_SIZE,BLOCK_SIZE);
    dim3 blocks1((nx-8+BLOCK_SIZE-1)/BLOCK_SIZE,(nz+BLOCK_SIZE-1)/BLOCK_SIZE);
    dim3 threads2(4,BLOCK_SIZE);
    dim3 blocks2(2,(nz+BLOCK_SIZE-1)/BLOCK_SIZE);

    cudaDx_interior<<<blocks1,threads1,0,streams[stream1]>>>(add, in, out, nx, nz, d);
    cudaKernelError();
    cudaDx_bnd<<<blocks2,threads2,0,streams[stream2]>>>(add, in, out, nx, nz, d);
    cudaKernelError();
}

void mult_Dz_gpu(bool add, const data_t* in, data_t* out, int nx, int nz, data_t d, const data_t* par, data_t a, int stream1, int stream2){
    
    dim3 threads1(BLOCK_SIZE,BLOCK_SIZE);
    dim3 blocks1((nx+BLOCK_SIZE-1)/BLOCK_SIZE,(nz-8+BLOCK_SIZE-1)/BLOCK_SIZE);
    dim3 threads2(BLOCK_SIZE,4);
    dim3 blocks2((nx+BLOCK_SIZE-1)/BLOCK_SIZE,2);

    cudaMultDz_interior<<<blocks1,threads1,0,streams[stream1]>>>(add, in, out, nx, nz, d, par, a);
    cudaKernelError();
    cudaMultDz_bnd<<<blocks2,threads2,0,streams[stream2]>>>(add, in, out, nx, nz, d, par, a);
    cudaKernelError();
}

void mult_Dx_gpu(bool add, const data_t* in, data_t* out, int nx, int nz, data_t d, const data_t * par, data_t a, int stream1, int stream2){

    dim3 threads1(BLOCK_SIZE,BLOCK_SIZE);
    dim3 blocks1((nx-8+BLOCK_SIZE-1)/BLOCK_SIZE,(nz+BLOCK_SIZE-1)/BLOCK_SIZE);
    dim3 threads2(4,BLOCK_SIZE);
    dim3 blocks2(2,(nz+BLOCK_SIZE-1)/BLOCK_SIZE);

    cudaMultDx_interior<<<blocks1,threads1,0,streams[stream1]>>>(add, in, out, nx, nz, d, par, a);
    cudaKernelError();
    cudaMultDx_bnd<<<blocks2,threads2,0,streams[stream2]>>>(add, in, out, nx, nz, d, par, a);
    cudaKernelError();
}

void Dzz_var_gpu(bool add, const data_t* in, data_t* out, int nx, int nz, data_t d, const data_t * par, data_t a, int stream1, int stream2){

    dim3 threads1(BLOCK_SIZE,BLOCK_SIZE);
    dim3 blocks1((nx+BLOCK_SIZE-1)/BLOCK_SIZE,(nz-12+BLOCK_SIZE-1)/BLOCK_SIZE);
    dim3 threads2(BLOCK_SIZE,6);
    dim3 blocks2((nx+BLOCK_SIZE-1)/BLOCK_SIZE,2);

    cudaDzz_var_interior<<<blocks1,threads1,0,streams[stream1]>>>(add, in, out, nx, nz, d, par, a);
    cudaKernelError();
    cudaDzz_var_bnd<<<blocks2,threads2,0,streams[stream2]>>>(add, in, out, nx, nz, d, par, a);
    cudaKernelError();
}

void Dxx_var_gpu(bool add, const data_t* in, data_t* out, int nx, int nz, data_t d, const data_t * par, data_t a, int stream1, int stream2){

    dim3 threads1(BLOCK_SIZE,BLOCK_SIZE);
    dim3 blocks1((nx-12+BLOCK_SIZE-1)/BLOCK_SIZE,(nz+BLOCK_SIZE-1)/BLOCK_SIZE);
    dim3 threads2(6,BLOCK_SIZE);
    dim3 blocks2(2,(nz+BLOCK_SIZE-1)/BLOCK_SIZE);

    cudaDxx_var_interior<<<blocks1,threads1,0,streams[stream1]>>>(add, in, out, nx, nz, d, par, a);
    cudaKernelError();
    cudaDxx_var_bnd<<<blocks2,threads2,0,streams[stream2]>>>(add, in, out, nx, nz, d, par, a);
    cudaKernelError();
}

void esat_neumann_top_gpu(bool add, const data_t* in0, const data_t*in1, data_t* out, int nx, int nz, data_t dx, data_t dz, const data_t * par0, const data_t * par1, data_t a0, data_t a1, int stream){
    
    dim3 threads(BLOCK_SIZE,1);
    dim3 blocks((nx+BLOCK_SIZE-1)/BLOCK_SIZE,1);

    cudaEsatNeumannTop<<<blocks,threads,0,streams[stream]>>>(add, in0, in1, out, nx, nz, dx, dz, par0, par1, a0, a1);
    cudaKernelError();
}
void esat_neumann_bottom_gpu(bool add, const data_t* in0, const data_t*in1, data_t* out, int nx, int nz, data_t dx, data_t dz, const data_t * par0, const data_t * par1, data_t a0, data_t a1, int stream){
    
    dim3 threads(BLOCK_SIZE,1);
    dim3 blocks((nx+BLOCK_SIZE-1)/BLOCK_SIZE,1);

    cudaEsatNeumannBottom<<<blocks,threads,0,streams[stream]>>>(add, in0, in1, out, nx, nz, dx, dz, par0, par1, a0, a1);
    cudaKernelError();
}
void esat_neumann_left_gpu(bool add, const data_t* in0, const data_t*in1, data_t* out, int nx, int nz, data_t dx, data_t dz, const data_t * par0, const data_t * par1, data_t a0, data_t a1, int stream){
    
    dim3 threads(1,BLOCK_SIZE);
    dim3 blocks(1,(nz+BLOCK_SIZE-1)/BLOCK_SIZE);

    cudaEsatNeumannLeft<<<blocks,threads,0,streams[stream]>>>(add, in0, in1, out, nx, nz, dx, dz, par0, par1, a0, a1);
    cudaKernelError();
}
void esat_neumann_right_gpu(bool add, const data_t* in0, const data_t*in1, data_t* out, int nx, int nz, data_t dx, data_t dz, const data_t * par0, const data_t * par1, data_t a0, data_t a1, int stream){
    
    dim3 threads(1,BLOCK_SIZE);
    dim3 blocks(1,(nz+BLOCK_SIZE-1)/BLOCK_SIZE);

    cudaEsatNeumannRight<<<blocks,threads,0,streams[stream]>>>(add, in0, in1, out, nx, nz, dx, dz, par0, par1, a0, a1);
    cudaKernelError();
}

void esat_absorbing_top_gpu(bool add, const data_t* in0, const data_t* in1, const data_t* in2, data_t* out, int nx, int nz, data_t dx, data_t dz, data_t dt, const data_t * par0,  const data_t * par1, const data_t * par2, data_t a0, data_t a1, data_t a2, int stream){
    
    dim3 threads(BLOCK_SIZE,1);
    dim3 blocks((nx+BLOCK_SIZE-1)/BLOCK_SIZE,1);

    cudaEsatAbsorbingTop<<<blocks,threads,0,streams[stream]>>>(add, in0, in1, in2, out, nx, nz, dx, dz, dt, par0, par1, par2, a0, a1, a2);
    cudaKernelError();
}
void esat_absorbing_bottom_gpu(bool add, const data_t* in0, const data_t* in1, const data_t* in2, data_t* out, int nx, int nz, data_t dx, data_t dz, data_t dt, const data_t * par0,  const data_t * par1, const data_t * par2, data_t a0, data_t a1, data_t a2, int stream){
    
    dim3 threads(BLOCK_SIZE,1);
    dim3 blocks((nx+BLOCK_SIZE-1)/BLOCK_SIZE,1);

    cudaEsatAbsorbingBottom<<<blocks,threads,0,streams[stream]>>>(add, in0, in1, in2, out, nx, nz, dx, dz, dt, par0, par1, par2, a0, a1, a2);
    cudaKernelError();
}
void esat_absorbing_left_gpu(bool add, const data_t* in0, const data_t* in1, const data_t* in2, data_t* out, int nx, int nz, data_t dx, data_t dz, data_t dt, const data_t * par0,  const data_t * par1, const data_t * par2, data_t a0, data_t a1, data_t a2, int stream){
    
    dim3 threads(1,BLOCK_SIZE);
    dim3 blocks(1,(nz+BLOCK_SIZE-1)/BLOCK_SIZE);

    cudaEsatAbsorbingLeft<<<blocks,threads,0,streams[stream]>>>(add, in0, in1, in2, out, nx, nz, dx, dz, dt, par0, par1, par2, a0, a1, a2);
    cudaKernelError();
}
void esat_absorbing_right_gpu(bool add, const data_t* in0, const data_t* in1, const data_t* in2, data_t* out, int nx, int nz, data_t dx, data_t dz, data_t dt, const data_t * par0,  const data_t * par1, const data_t * par2, data_t a0, data_t a1, data_t a2, int stream){
    
    dim3 threads(1,BLOCK_SIZE);
    dim3 blocks(1,(nz+BLOCK_SIZE-1)/BLOCK_SIZE);

    cudaEsatAbsorbingRight<<<blocks,threads,0,streams[stream]>>>(add, in0, in1, in2, out, nx, nz, dx, dz, dt, par0, par1, par2, a0, a1, a2);
    cudaKernelError();
}

void esat_Dz_top_gpu(bool add, const data_t * in, data_t * out, int nx, int nz, data_t d, const data_t * par, data_t a, int stream){ 
    dim3 threads(BLOCK_SIZE,1);
    dim3 blocks((nx+BLOCK_SIZE-1)/BLOCK_SIZE,1);

    cudaDzTop<<<blocks,threads,0,streams[stream]>>>(add, in, out, nx, nz, d, par, a);
    cudaKernelError();
}
void esat_Dz_bottom_gpu(bool add, const data_t * in, data_t * out, int nx, int nz, data_t d, const data_t * par, data_t a, int stream){
    dim3 threads(BLOCK_SIZE,1);
    dim3 blocks((nx+BLOCK_SIZE-1)/BLOCK_SIZE,1);

    cudaDzBottom<<<blocks,threads,0,streams[stream]>>>(add, in, out, nx, nz, d, par, a);
    cudaKernelError();
}
void esat_Dx_left_gpu(bool add, const data_t * in, data_t * out, int nx, int nz, data_t d, const data_t * par, data_t a, int stream){
    dim3 threads(1,BLOCK_SIZE);
    dim3 blocks(1,(nz+BLOCK_SIZE-1)/BLOCK_SIZE);

    cudaDxLeft<<<blocks,threads,0,streams[stream]>>>(add, in, out, nx, nz, d, par, a);
    cudaKernelError();
}
void esat_Dx_right_gpu(bool add, const data_t * in, data_t * out, int nx, int nz, data_t d, const data_t * par, data_t a, int stream){
    dim3 threads(1,BLOCK_SIZE);
    dim3 blocks(1,(nz+BLOCK_SIZE-1)/BLOCK_SIZE);

    cudaDxRight<<<blocks,threads,0,streams[stream]>>>(add, in, out, nx, nz, d, par, a);
    cudaKernelError();
}

void esat_scale_boundaries_gpu(data_t* in, int nx, int nz, data_t dx, data_t dz, const data_t* par, data_t dt, bool top, bool bottom, bool left, bool right, int stream1, int stream2){
    
    dim3 threads1(BLOCK_SIZE,1); // top-bottom excluding the corners
    dim3 blocks1((nx-2+BLOCK_SIZE-1)/BLOCK_SIZE,2);
    dim3 threads2(1,BLOCK_SIZE); // left-right 
    dim3 blocks2(2,(nz+BLOCK_SIZE-1)/BLOCK_SIZE);
    
    if  (top || bottom) {
        cudaScaleTopBottom<<<blocks1,threads1,0,streams[stream1]>>>(in, nx, nz, dx, dz, par, dt, top, bottom);
        cudaKernelError();
    }
    if  (top || bottom || left || right) {
        cudaScaleLeftRight<<<blocks2,threads2,0,streams[stream2]>>>(in, nx, nz, dx, dz, par, dt, top, bottom, left, right);
        cudaKernelError();
    }
}
void vtisat_scale_boundaries_gpu(data_t* in, int nx, int nz, data_t dx, data_t dz, const data_t* par, data_t dt, bool top, bool bottom, bool left, bool right, int stream1, int stream2){

    dim3 threads1(BLOCK_SIZE,1); // top-bottom excluding the corners
    dim3 blocks1((nx-2+BLOCK_SIZE-1)/BLOCK_SIZE,2);
    dim3 threads2(1,BLOCK_SIZE); // left-right 
    dim3 blocks2(2,(nz+BLOCK_SIZE-1)/BLOCK_SIZE);
    
    if  (top || bottom) {
        cudaScaleTopBottom<<<blocks1,threads1,0,streams[stream1]>>>(in, nx, nz, dx, dz, par, dt, top, bottom);
        cudaKernelError();
    }
    if  (top || bottom || left || right) {
        cudaScaleLeftRightVTI<<<blocks2,threads2,0,streams[stream2]>>>(in, nx, nz, dx, dz, par, dt, top, bottom, left, right);
        cudaKernelError();
    }
}

void taper_top_gpu(data_t* in, int nx, int nz, int taper, data_t a, int stream){

    dim3 threads(BLOCK_SIZE/2,taper);
    dim3 blocks(2*(nx+BLOCK_SIZE/2-1)/BLOCK_SIZE,1);
    cudaTaperTop<<<blocks,threads,0,streams[stream]>>>(in,nx,nz,taper,a);
    cudaKernelError();
}

void taper_bottom_gpu(data_t* in, int nx, int nz, int taper, data_t a, int stream){

    dim3 threads(BLOCK_SIZE/2,taper);
    dim3 blocks(2*(nx+BLOCK_SIZE/2-1)/BLOCK_SIZE,1);
    cudaTaperBottom<<<blocks,threads,0,streams[stream]>>>(in,nx,nz,taper,a);
    cudaKernelError();
}

void taper_left_gpu(data_t* in, int nx, int nz, int taper, data_t a, int stream){

    dim3 threads(taper,BLOCK_SIZE/2);
    dim3 blocks(1,2*(nx+BLOCK_SIZE/2-1)/BLOCK_SIZE);
    cudaTaperLeft<<<blocks,threads,0,streams[stream]>>>(in,nx,nz,taper,a);
    cudaKernelError();
}

void taper_right_gpu(data_t* in, int nx, int nz, int taper, data_t a, int stream){

    dim3 threads(taper,BLOCK_SIZE/2);
    dim3 blocks(1,2*(nx+BLOCK_SIZE/2-1)/BLOCK_SIZE);
    cudaTaperRight<<<blocks,threads,0,streams[stream]>>>(in,nx,nz,taper,a);
    cudaKernelError();
}

void time_step_gpu(const data_t * prev, const data_t * curr, data_t * next, const data_t * par, int nx, int nz, data_t dt, int stream){

    dim3 threads(BLOCK_SIZE,BLOCK_SIZE);
    dim3 blocks((nx+BLOCK_SIZE-1)/BLOCK_SIZE,(nz+BLOCK_SIZE-1)/BLOCK_SIZE);
    cudaTimeStep<<<blocks,threads,0,streams[stream]>>>(prev,curr,next,par,nx,nz,dt);
    cudaKernelError();
}

void delta_m3::inject_gpu(bool add, const data_t ** in, data_t * out, int nx, int nz, int nt, int ntr, int it, int itr_min, int itr_max, const int * xind, const int * zind, const data_t * xw, const data_t * zw) const{

    if (add==false) hipMemset(out,0,nx*nz*sizeof(data_t));
    dim3 threads(3,3);
    dim3 blocks(itr_max-itr_min,1);
    cudaInjectDM3<<<blocks,threads,0,streams[1]>>>(in, out, nx, nz, nt, ntr, it, itr_min, itr_max, xind, zind, xw, zw);
    cudaKernelError();
}

void delta_m3::extract_gpu(bool add, const data_t * in, data_t ** out, int nx, int nz, int nt, int ntr, int it, int itr_min, int itr_max, const int * xind, const int * zind, const data_t * xw, const data_t * zw) const{

    if (add==false) hipMemset(out[0]+itr_min,0,nt*(itr_max-itr_min)*sizeof(data_t));
    dim3 threads(3,3);
    dim3 blocks(itr_max-itr_min,1);
    cudaExtractDM3<<<blocks,threads,0,streams[1]>>>(in, out, nx, nz, nt, ntr, it, itr_min, itr_max, xind, zind, xw, zw);
    cudaKernelError();
}

void ddelta_m3::inject_gpu(bool add, const data_t ** in, data_t * out, int nx, int nz, int nt, int ntr, int it, int itr_min, int itr_max, const int * xind, const int * zind, const data_t * xw, const data_t * zw) const{

    if (add==false) hipMemset(out,0,nx*nz*sizeof(data_t));
    dim3 threads(3,3);
    dim3 blocks(itr_max-itr_min,1);
    cudaInjectDDM3<<<blocks,threads,0,streams[1]>>>(in, out, nx, nz, nt, ntr, it, itr_min, itr_max, xind, zind, xw, zw);
    cudaKernelError();
}

void ddelta_m3::extract_gpu(bool add, const data_t * in, data_t ** out, int nx, int nz, int nt, int ntr, int it, int itr_min, int itr_max, const int * xind, const int * zind, const data_t * xw, const data_t * zw) const{

    if (add==false) {
        hipMemset(out[0]+itr_min,0,nt*(itr_max-itr_min)*sizeof(data_t));
        hipMemset(out[1]+itr_min,0,nt*(itr_max-itr_min)*sizeof(data_t));
    }
    dim3 threads(3,3);
    dim3 blocks(itr_max-itr_min,1);
    cudaExtractDDM3<<<blocks,threads,0,streams[1]>>>(in, out, nx, nz, nt, ntr, it, itr_min, itr_max, xind, zind, xw, zw);
    cudaKernelError();
}

void dipole_m3::inject_gpu(bool add, const data_t ** in, data_t * out, int nx, int nz, int nt, int ntr, int it, int itr_min, int itr_max, const int * xind, const int * zind, const data_t * xw, const data_t * zw) const{

    if (add==false) hipMemset(out,0,nx*nz*sizeof(data_t));
    dim3 threads(3,3);
    dim3 blocks(itr_max-itr_min,1);
    cudaInjectDIPM3<<<blocks,threads,0,streams[1]>>>(in, out, nx, nz, nt, ntr, it, itr_min, itr_max, xind, zind, xw, zw);
    cudaKernelError();
}

void dipole_m3::extract_gpu(bool add, const data_t * in, data_t ** out, int nx, int nz, int nt, int ntr, int it, int itr_min, int itr_max, const int * xind, const int * zind, const data_t * xw, const data_t * zw) const{

    if (add==false) hipMemset(out[0]+itr_min,0,nt*(itr_max-itr_min)*sizeof(data_t));
    dim3 threads(3,3);
    dim3 blocks(itr_max-itr_min,1);
    cudaExtractDIPM3<<<blocks,threads,0,streams[1]>>>(in, out, nx, nz, nt, ntr, it, itr_min, itr_max, xind, zind, xw, zw);
    cudaKernelError();
}

void nl_we_op_e::compute_gradients_gpu(const data_t * model, const data_t * u_for, const data_t * curr, const data_t * u_x, const data_t * u_z, data_t * tmp, data_t * grad, const param &par, int nx, int nz, int it, data_t dx, data_t dz, data_t dt) const {
    
    int nxz = nx*nz;
    const data_t * pfor1x=u_for+2*nxz;
    const data_t * pfor1z=u_for+3*nxz;

    Dx_gpu(false, pfor1x, tmp, nx, nz, dx, 3, 4); // forwardx_x
    Dz_gpu(false, pfor1z, tmp+nxz, nx, nz, dz, 4, 3); // forwardz_z
    Dx_gpu(false, pfor1z, tmp+2*nxz, nx, nz, dx, 3, 4); // forwardz_x
    Dz_gpu(false, pfor1x, tmp+3*nxz, nx, nz, dz, 4, 3); // forwardx_z
    hipStreamSynchronize(streams[3]);
    hipStreamSynchronize(streams[4]);

    dim3 threads(BLOCK_SIZE,BLOCK_SIZE);
    dim3 blocks((nx+BLOCK_SIZE-1)/BLOCK_SIZE,(nz+BLOCK_SIZE-1)/BLOCK_SIZE);
    cudaComputeGradients<<<blocks,threads,0,streams[3]>>>(model, u_for, curr, u_x, u_z, tmp, grad, nx, nz, par.nt, par.sub, it, dx, dz, dt);
    cudaKernelError();
}

void nl_we_op_vti::compute_gradients_gpu(const data_t * model, const data_t * u_for, const data_t * curr, const data_t * u_x, const data_t * u_z, data_t * tmp, data_t * grad, const param &par, int nx, int nz, int it, data_t dx, data_t dz, data_t dt) const {
    
    int nxz = nx*nz;
    const data_t * pfor1x=u_for+2*nxz;
    const data_t * pfor1z=u_for+3*nxz;

    Dx_gpu(false, pfor1x, tmp, nx, nz, dx, 3, 4); // forwardx_x
    Dz_gpu(false, pfor1z, tmp+nxz, nx, nz, dz, 4, 3); // forwardz_z
    Dx_gpu(false, pfor1z, tmp+2*nxz, nx, nz, dx, 3, 4); // forwardz_x
    Dz_gpu(false, pfor1x, tmp+3*nxz, nx, nz, dz, 4, 3); // forwardx_z
    hipStreamSynchronize(streams[3]);
    hipStreamSynchronize(streams[4]);

    dim3 threads(BLOCK_SIZE,BLOCK_SIZE);
    dim3 blocks((nx+BLOCK_SIZE-1)/BLOCK_SIZE,(nz+BLOCK_SIZE-1)/BLOCK_SIZE);
    cudaComputeGradientsVTI<<<blocks,threads,0,streams[3]>>>(model, u_for, curr, u_x, u_z, tmp, grad, nx, nz, par.nt, par.sub, it, dx, dz, dt);
    cudaKernelError();
}


#undef BLOCK_SIZE